#include "hip/hip_runtime.h"
#include "defines.h"
#include "diff.h"
#include "gaxtch.h"

#include <cstdio>
#include <cstring>

str_res test_gaxtch( const UIN cudaBlockSize, const str_matAXT matAXT, const FPT * ref )
{
	const UIN th           = matAXT.tileH;
	char TH[5]; sprintf( TH, "%d", th );
	char buffer[48];
	strcpy( buffer, "gaxtch" );
	strcat( buffer, TH );
	str_res sr;
	strcpy( sr.name, buffer );
	if ( ( (strcmp(matAXT.name, "M24_circuit5M.bin") == 0) && (cudaBlockSize == 1024) ) || ( (strcmp(matAXT.name, "M23_delaunay_n23.bin") == 0) && (cudaBlockSize == 1024) ) )
	{
		sr.et        = 0.0;
		sr.ot        = 0.0;
		sr.flops     = 0.0;
		sr.sErr.aErr = 0.0;
		sr.sErr.rErr = 0.0;
		sr.sErr.pos  = 0;
	}
	else
	{
		// 
		const UIN tn           = matAXT.tileN;
		const UIN thw          = matAXT.tileHW;
		const UIN log          = matAXT.log;
		const UIN cudaBlockNum = ( (tn*32) + cudaBlockSize - 1 ) / cudaBlockSize;
		const UIN wpb          = cudaBlockSize / 32;
		const UIN devLenAX     = cudaBlockNum * 2 * th * thw * wpb;
		const UIN devLenSEC    = cudaBlockNum     * th * thw * wpb;
		// allocate memory on GPU
		FPT * d_ax;  HANDLE_CUDA_ERROR( hipMalloc( &d_ax,  devLenAX      * sizeof(FPT) ) ); TEST_POINTER( d_ax  );
		UIN * d_hdr; HANDLE_CUDA_ERROR( hipMalloc( &d_hdr, devLenSEC     * sizeof(UIN) ) ); TEST_POINTER( d_hdr );
		FPT * d_res; HANDLE_CUDA_ERROR( hipMalloc( &d_res, matAXT.nrows  * sizeof(FPT) ) ); TEST_POINTER( d_res );
		// copy necessary arrays to device
		HANDLE_CUDA_ERROR( hipMemset( d_ax,  0, devLenAX  * sizeof(FPT) ) );
		HANDLE_CUDA_ERROR( hipMemset( d_hdr, 0, devLenSEC * sizeof(UIN) ) );
		HANDLE_CUDA_ERROR( hipMemcpy( d_ax,  matAXT.ax,  matAXT.lenAX  * sizeof(FPT), hipMemcpyHostToDevice ) );
		HANDLE_CUDA_ERROR( hipMemcpy( d_hdr, matAXT.sec, matAXT.lenSEC * sizeof(UIN), hipMemcpyHostToDevice ) );
		// create events for time measuring
		hipEvent_t cet1; HANDLE_CUDA_ERROR( hipEventCreate( &cet1 ) );
		hipEvent_t cet2; HANDLE_CUDA_ERROR( hipEventCreate( &cet2 ) );
		// timed iterations
		float ti = 0.0f, tt = 0.0f;
		UIN i;
		for ( i = 0; i < NUM_ITE; i++ )
		{
			HANDLE_CUDA_ERROR( hipMemset( d_res, 0, matAXT.nrows  * sizeof(FPT) ) );
			HANDLE_CUDA_ERROR( hipEventRecord( cet1 ) );
			gaxtch <<<cudaBlockNum, cudaBlockSize>>> ( log, th, d_ax, d_hdr, d_res );
			HANDLE_CUDA_ERROR( hipEventRecord( cet2 ) );
			HANDLE_CUDA_ERROR( hipEventSynchronize( cet2 ) );
			HANDLE_CUDA_ERROR( hipEventElapsedTime( &ti, cet1, cet2 ) );
			tt = tt + ti;
		}
		// destroy events for time measuring
		HANDLE_CUDA_ERROR( hipEventDestroy( cet1 ) );
		HANDLE_CUDA_ERROR( hipEventDestroy( cet2 ) );
		// copy result from device
		FPT * res = (FPT *) malloc( matAXT.nrows * sizeof(FPT) ); TEST_POINTER( res );
		HANDLE_CUDA_ERROR( hipMemcpy( res, d_res, matAXT.nrows * sizeof(FPT), hipMemcpyDeviceToHost ) );
		// free device memory
		HANDLE_CUDA_ERROR( hipFree( d_ax  ) );
		HANDLE_CUDA_ERROR( hipFree( d_hdr ) );
		HANDLE_CUDA_ERROR( hipFree( d_res ) );
		// store results
		sr.et    = ( (double) tt / (double) NUM_ITE ) * 1e-3;
		sr.ot    = 0.0;
		sr.flops = ( 2.0 * ( (double) matAXT.nnz ) ) / sr.et;
		get_errors( matAXT.nrows, ref, res, &(sr.sErr) );
		// free cpu memory
		free( res );
	}
	return( sr );
}

