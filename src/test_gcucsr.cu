#include "defines.h"
#include "diff.h"

#include <cstdio>
#include <cstring>
#include <hipsparse.h>

#define HANDLE_CUSPARSE_ERROR( cseID ) { if ( cseID != HIPSPARSE_STATUS_SUCCESS ) { printf( "FILE: %s LINE: %d CUBLAS_ERROR: %s\n", __FILE__, __LINE__, cusparseGetErrorMessage( cseID ) ); printf( "\nvim %s +%d\n", __FILE__, __LINE__); exit( EXIT_FAILURE ); } }

static __host__ const char * cusparseGetErrorMessage( hipsparseStatus_t statusID )
{
        switch(statusID)
        {
                case HIPSPARSE_STATUS_NOT_INITIALIZED:           return "HIPSPARSE_STATUS_NOT_INITIALIZED";
                case HIPSPARSE_STATUS_ALLOC_FAILED:              return "HIPSPARSE_STATUS_ALLOC_FAILED";
                case HIPSPARSE_STATUS_INVALID_VALUE:             return "HIPSPARSE_STATUS_INVALID_VALUE";
                case HIPSPARSE_STATUS_ARCH_MISMATCH:             return "HIPSPARSE_STATUS_ARCH_MISMATCH";
                case HIPSPARSE_STATUS_MAPPING_ERROR:             return "HIPSPARSE_STATUS_MAPPING_ERROR";
                case HIPSPARSE_STATUS_EXECUTION_FAILED:          return "HIPSPARSE_STATUS_EXECUTION_FAILED";
                case HIPSPARSE_STATUS_INTERNAL_ERROR:            return "HIPSPARSE_STATUS_INTERNAL_ERROR";
                case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
        }
        return "<cusparse unknown>";
}

str_res test_gcucsr( const str_matCSR matCSR, const FPT * vec, const FPT * ref )
{
	// get parameteres for cuSPARSE
	const UIN                     nrows = matCSR.nrows;
	const UIN                       nnz = matCSR.nnz;
	hipsparseHandle_t    cusparseH = NULL;
	const hipsparseSpMVAlg_t cusparseAM = HIPSPARSE_SPMV_CSR_ALG1;
	const hipsparseOperation_t cusparseO = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseSpMatDescr_t cusparseMD = NULL;
	hipsparseDnVecDescr_t cusparseVD1 = NULL, cusparseVD2 = NULL;
	size_t    cudaSpaceBufferSize;
	const FPT                      zero = (FPT)  0;
	const FPT                       one = (FPT)  1;
	#if FP_TYPE == FP_FLOAT
		hipDataType cudaDT = HIP_R_32F;
	#else
		hipDataType cudaDT = HIP_R_64F;
	#endif
	// allocate memory on GPU
	FPT * d_val; HANDLE_CUDA_ERROR( hipMalloc( &d_val,           nnz * sizeof(FPT) ) ); TEST_POINTER( d_val );
	int * d_col; HANDLE_CUDA_ERROR( hipMalloc( &d_col,           nnz * sizeof(int) ) ); TEST_POINTER( d_col );
	int * d_row; HANDLE_CUDA_ERROR( hipMalloc( &d_row, ( nrows + 1 ) * sizeof(int) ) ); TEST_POINTER( d_row );
	FPT * d_vec; HANDLE_CUDA_ERROR( hipMalloc( &d_vec,         nrows * sizeof(FPT) ) ); TEST_POINTER( d_vec );
	FPT * d_res; HANDLE_CUDA_ERROR( hipMalloc( &d_res,         nrows * sizeof(FPT) ) ); TEST_POINTER( d_res );
	// copy necessary arrays to device
	HANDLE_CUDA_ERROR( hipMemcpy( d_val, matCSR.val,           nnz * sizeof(FPT), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_col, matCSR.col,           nnz * sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_row, matCSR.row, ( nrows + 1 ) * sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_vec, vec,                nrows * sizeof(FPT), hipMemcpyHostToDevice ) );
        // create handlers for cuSPARSE
        HANDLE_CUSPARSE_ERROR( hipsparseCreate(&cusparseH) );
        HANDLE_CUSPARSE_ERROR( hipsparseCreateCsr(&cusparseMD, matCSR.nrows, matCSR.nrows, matCSR.nnz,
                                      (void *)d_row, (void *)d_col, (void *)d_val,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, cudaDT ));
        HANDLE_CUSPARSE_ERROR( hipsparseCreateDnVec(&cusparseVD1, matCSR.nrows, (void*)d_vec, cudaDT ));
        HANDLE_CUSPARSE_ERROR( hipsparseCreateDnVec(&cusparseVD2, matCSR.nrows, d_res, cudaDT ));
	// get space buffer for hipsparseSpMV
	HANDLE_CUSPARSE_ERROR( hipsparseSpMV_bufferSize(cusparseH, cusparseO,
                    (void *)&one, cusparseMD, cusparseVD1, (void *)&zero,
                    cusparseVD2, cudaDT, cusparseAM, &cudaSpaceBufferSize ));
	void * cudaSpaceBuffer; HANDLE_CUDA_ERROR( hipMalloc( &cudaSpaceBuffer, cudaSpaceBufferSize ) );
	// create events for time measuring
	hipEvent_t cet1; HANDLE_CUDA_ERROR( hipEventCreate( &cet1 ) );
	hipEvent_t cet2; HANDLE_CUDA_ERROR( hipEventCreate( &cet2 ) );
	// timed iterations
	float ti = 0.0, tt = 0.0;
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		HANDLE_CUDA_ERROR( hipEventRecord( cet1 ) );
		HANDLE_CUSPARSE_ERROR( hipsparseSpMV( cusparseH, cusparseO,
                      (void *)&one, cusparseMD, cusparseVD1, (void *)&zero,
                      cusparseVD2, cudaDT, cusparseAM, cudaSpaceBuffer));
		HANDLE_CUDA_ERROR( hipEventRecord( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventSynchronize( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventElapsedTime( &ti, cet1, cet2 ) );
		tt = tt + ti;
	}
	// destroy events for time measuring
	HANDLE_CUDA_ERROR( hipEventDestroy( cet1 ) );
	HANDLE_CUDA_ERROR( hipEventDestroy( cet2 ) );
	// copy result from device
	FPT * res = (FPT *) calloc( nrows, sizeof(FPT) ); TEST_POINTER( res );
	HANDLE_CUDA_ERROR( hipMemcpy( res, d_res, nrows * sizeof(FPT), hipMemcpyDeviceToHost ) );
	// free device memory
	HANDLE_CUDA_ERROR( hipFree( d_val ) );
	HANDLE_CUDA_ERROR( hipFree( d_col ) );
	HANDLE_CUDA_ERROR( hipFree( d_row ) );
	HANDLE_CUDA_ERROR( hipFree( d_vec ) );
	HANDLE_CUDA_ERROR( hipFree( d_res ) );
	// store results
	str_res sr;
	strcpy( sr.name, "gcucsr" );
	sr.et    = ( (double) tt / (double) NUM_ITE ) * 1e-3;
	sr.ot    = 0.0;
	sr.flops = ( (double) matCSR.nnz * 2.0 ) / sr.et;
	get_errors( matCSR.nrows, ref, res, &(sr.sErr) );
	// free cpu memory
	free( res );
	return( sr );
}

