#include "defines.h"
#include "diff.h"
#include "gk1.h"
#include "k1.h"

#include <cstdio>
#include <cstring>

str_res test_gk1( const UIN cudaBlockSize, const str_matK1 matK1, const FPT * vec, const FPT * ref )
{
	// 
	UIN cudaBlockNum = ( matK1.nrows + cudaBlockSize - 1 ) / cudaBlockSize;
	// allocate memory on GPU
	FPT * d_val;   HANDLE_CUDA_ERROR( hipMalloc( &d_val,   matK1.lenVC                 * sizeof(FPT) ) ); TEST_POINTER( d_val   );
	UIN * d_col;   HANDLE_CUDA_ERROR( hipMalloc( &d_col,   matK1.lenVC                 * sizeof(UIN) ) ); TEST_POINTER( d_col   );
	UIN * d_nmc;   HANDLE_CUDA_ERROR( hipMalloc( &d_nmc,   matK1.chunkNum              * sizeof(UIN) ) ); TEST_POINTER( d_nmc   );
	UIN * d_chp;   HANDLE_CUDA_ERROR( hipMalloc( &d_chp,   matK1.chunkNum              * sizeof(UIN) ) ); TEST_POINTER( d_chp   );
	UIN * d_permi; HANDLE_CUDA_ERROR( hipMalloc( &d_permi, matK1.chunkNum * CHUNK_SIZE * sizeof(UIN) ) ); TEST_POINTER( d_permi );
	FPT * d_vec;   HANDLE_CUDA_ERROR( hipMalloc( &d_vec,   matK1.nrows                 * sizeof(FPT) ) ); TEST_POINTER( d_vec   );
	FPT * d_res;   HANDLE_CUDA_ERROR( hipMalloc( &d_res,   matK1.nrows                 * sizeof(FPT) ) ); TEST_POINTER( d_res   );
	// copy necessary arrays to device
	HANDLE_CUDA_ERROR( hipMemcpy( d_val,   matK1.val,   matK1.lenVC                 * sizeof(FPT), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_col,   matK1.col,   matK1.lenVC                 * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_nmc,   matK1.nmc,   matK1.chunkNum              * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_chp,   matK1.chp,   matK1.chunkNum              * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_permi, matK1.permi, matK1.chunkNum * CHUNK_SIZE * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_vec,   vec,         matK1.nrows                 * sizeof(FPT), hipMemcpyHostToDevice ) );
	// create events for time measuring
	hipEvent_t cet1; HANDLE_CUDA_ERROR( hipEventCreate( &cet1 ) );
	hipEvent_t cet2; HANDLE_CUDA_ERROR( hipEventCreate( &cet2 ) );
	// timed iterations
	float ti = 0.0f, tt = 0.0f;
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		HANDLE_CUDA_ERROR( hipEventRecord( cet1 ) );
		gk1 <<<cudaBlockNum, cudaBlockSize>>> (  matK1.nrows, d_val, d_col, d_nmc, d_chp, d_permi, d_vec, d_res );
		HANDLE_CUDA_ERROR( hipEventRecord( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventSynchronize( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventElapsedTime( &ti, cet1, cet2 ) );
		tt = tt + ti;
	}
	// destroy events for time measuring
	HANDLE_CUDA_ERROR( hipEventDestroy( cet1 ) );
	HANDLE_CUDA_ERROR( hipEventDestroy( cet2 ) );
	// copy result from device
	FPT * res = (FPT *) malloc( matK1.nrows * sizeof(FPT) ); TEST_POINTER( res );
	HANDLE_CUDA_ERROR( hipMemcpy( res, d_res, matK1.nrows * sizeof(FPT), hipMemcpyDeviceToHost ) );
	// free device memory
	HANDLE_CUDA_ERROR( hipFree( d_val   ) );
	HANDLE_CUDA_ERROR( hipFree( d_col   ) );
	HANDLE_CUDA_ERROR( hipFree( d_nmc   ) );
	HANDLE_CUDA_ERROR( hipFree( d_chp   ) );
	HANDLE_CUDA_ERROR( hipFree( d_permi ) );
	HANDLE_CUDA_ERROR( hipFree( d_vec   ) );
	HANDLE_CUDA_ERROR( hipFree( d_res   ) );
	// store results
	str_res sr;
	strcpy( sr.name, "gk1" );
	sr.et    = ( (double) tt / (double) NUM_ITE ) * 1e-3;
	sr.ot    = 0.0;
	sr.flops = ( 2.0 * ( (double) matK1.nnz ) ) / sr.et;
	get_errors( matK1.nrows, ref, res, &(sr.sErr) );
	// free cpu memory
	free( res );
	return( sr );
}

