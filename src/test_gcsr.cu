#include "defines.h"
#include "diff.h"
#include "gcsr.h"

#include <cstdio>
#include <cstring>

str_res test_gcsr( const UIN cudaBlockSize, const str_matCSR matCSR, const FPT * vec, const FPT * ref )
{
	// get parameters
	const UIN        nrows = matCSR.nrows;
	const UIN          nnz = matCSR.nnz;
	const UIN cudaBlockNum = ( nrows + cudaBlockSize - 1 ) / cudaBlockSize;
	// allocate memory on GPU
	FPT * d_val; HANDLE_CUDA_ERROR( hipMalloc( &d_val,          nnz * sizeof(FPT) ) ); TEST_POINTER( d_val );
	UIN * d_col; HANDLE_CUDA_ERROR( hipMalloc( &d_col,          nnz * sizeof(UIN) ) ); TEST_POINTER( d_col );
	UIN * d_row; HANDLE_CUDA_ERROR( hipMalloc( &d_row, (nrows + 1 ) * sizeof(UIN) ) ); TEST_POINTER( d_row );
	FPT * d_vec; HANDLE_CUDA_ERROR( hipMalloc( &d_vec,        nrows * sizeof(FPT) ) ); TEST_POINTER( d_vec );
	FPT * d_res; HANDLE_CUDA_ERROR( hipMalloc( &d_res,        nrows * sizeof(FPT) ) ); TEST_POINTER( d_res );
	// copy necessary arrays to device
	HANDLE_CUDA_ERROR( hipMemcpy( d_val, matCSR.val,          nnz * sizeof(FPT), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_col, matCSR.col,          nnz * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_row, matCSR.row, ( nrows + 1 )* sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_vec, vec,               nrows * sizeof(FPT), hipMemcpyHostToDevice ) );
	// create events for time measuring
	hipEvent_t cet1; HANDLE_CUDA_ERROR( hipEventCreate( &cet1 ) );
	hipEvent_t cet2; HANDLE_CUDA_ERROR( hipEventCreate( &cet2 ) );
	// timed iterations
	float ti = 0.0f, tt = 0.0f;
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		HANDLE_CUDA_ERROR( hipEventRecord( cet1 ) );
		gcsr <<<cudaBlockNum, cudaBlockSize>>> ( nrows, d_val, d_col, d_row, d_vec, d_res );
		HANDLE_CUDA_ERROR( hipEventRecord( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventSynchronize( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventElapsedTime( &ti, cet1, cet2 ) );
		tt = tt + ti;
	}
	// destroy events for time measuring
	HANDLE_CUDA_ERROR( hipEventDestroy( cet1 ) );
	HANDLE_CUDA_ERROR( hipEventDestroy( cet2 ) );
	// copy result from device
	FPT * res = (FPT *) calloc( matCSR.nrows, sizeof(FPT) ); TEST_POINTER( res );
	HANDLE_CUDA_ERROR( hipMemcpy( res, d_res, nrows * sizeof(FPT), hipMemcpyDeviceToHost ) );
	// free device memory
	HANDLE_CUDA_ERROR( hipFree( d_val ) );
	HANDLE_CUDA_ERROR( hipFree( d_col ) );
	HANDLE_CUDA_ERROR( hipFree( d_row ) );
	HANDLE_CUDA_ERROR( hipFree( d_vec ) );
	HANDLE_CUDA_ERROR( hipFree( d_res ) );
	// store results
	str_res sr;
	strcpy( sr.name, "gcsr" );
	sr.et    = ( (double) tt / (double) NUM_ITE ) * 1e-3;
	sr.ot    = 0.0;
	sr.flops = ( 2.0 * ( (double) matCSR.nnz ) ) / sr.et;
	get_errors( matCSR.nrows, ref, res, &(sr.sErr) );
	// free cpu memory
	free( res );
	return( sr );
}

