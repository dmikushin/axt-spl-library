#include "hip/hip_runtime.h"
// ┌────────────────────────────────┐
// │program: cudaSpmv.cu            │
// │author: Edoardo Coronado        │
// │date: 21-08-2019 (dd-mm-yyyy)   │
// ╰────────────────────────────────┘



#ifndef __CUDA_SPMV_HEADER__
#define __CUDA_SPMV_HEADER__



#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hipsparse.h>
#ifdef _OMP_
	#include <omp.h>
	#ifndef OMP_SCH
		#define OMP_SCH static
		char omp_schedule[7] = "static";
	#endif
#endif



#ifndef FP_FLOAT
	#define FP_FLOAT  1
#endif



#ifndef FP_DOUBLE
	#define FP_DOUBLE 2
#endif



#if FP_TYPE == FP_FLOAT
	typedef float  FPT;
	char fptMsg[6] = "float";
#endif



#if FP_TYPE == FP_DOUBLE
	typedef double FPT;
	char fptMsg[7] = "double";
#endif



#ifndef UIN
	typedef unsigned int UIN;
#endif



#ifndef HDL
	#define HDL { fflush(stdout); printf( "---------------------------------------------------------------------------------------------------------\n" ); fflush(stdout); }
#endif



#ifndef BM
	#define BM { fflush(stdout); printf( "\nFile: %s    Line: %d.\n", __FILE__, __LINE__ ); fflush(stdout); }
#endif



#ifndef NUM_ITE
	#define NUM_ITE 250
#endif



#ifndef CHUNK_SIZE
	#define CHUNK_SIZE 32
#endif



typedef struct { UIN cbs; char matFileName[48]; UIN ompMT; } str_inputArgs;



static str_inputArgs checkArgs( const UIN argc, char ** argv )
{
	if ( argc < 3 )
	{
		fflush(stdout);
		printf( "\n\tMissing input arguments.\n" );
		printf( "\n\tUsage:\n\n\t\t%s <cudaBlockSize> <matFileName>\n\n", argv[0] );
		printf( "\t\t\t<cudaBlockSize>:  number of threads per cuda block.\n" );
		printf( "\t\t\t<matFileName>:    file's name that contains the matrix in CSR format [string].\n" );
		fflush(stdout);
		exit( EXIT_FAILURE );
	}
	str_inputArgs sia;
	sia.cbs    = atoi( argv[1] );
	strcpy( sia.matFileName, argv[2] );
	sia.ompMT = 1;
	#pragma omp parallel if(_OPENMP)
	{
		#pragma omp master
		{
			sia.ompMT = omp_get_max_threads();
		}
	}
	return( sia );
}



#ifndef ABORT
	#define ABORT { fflush(stdout); printf( "\nFile: %s Line: %d execution is aborted.\n", __FILE__, __LINE__ ); fflush(stdout); exit( EXIT_FAILURE ); }
#endif



static void printRunSettings( const str_inputArgs sia )
{
	FILE * fh = fopen( "HASH.txt", "r+" );
	char hash[128];
	if ( fscanf( fh, "%s", &(hash) ) != 1 ) ABORT;
	fclose( fh );
	time_t t = time(NULL);
	struct tm tm = *localtime(&t);
	HDL; printf( "run settings\n" ); HDL;
	#ifdef _CIRRUS_
	printf( "hostname:           %s\n", "cirrus.EPCC" );
	#endif
	#ifdef _KAY_
	printf( "hostname:           %s\n", "kay.ICHEC" );
	#endif
	#ifdef _CTGPGPU2_
	printf( "hostname:           %s\n", "ctgpgpu2.CITIUS" );
	#endif
	printf( "srcFileName:        %s\n", __FILE__ );
	printf( "gitHash:            %s\n", hash );
	printf( "date:               %d-%d-%d (yyyy-mm-dd)\n", tm.tm_year + 1900, tm.tm_mon + 1, tm.tm_mday );
	printf( "time:               %d:%d:%d (hh:mm:ss)\n", tm.tm_hour, tm.tm_min, tm.tm_sec );
	printf( "matFileName:        %s\n", sia.matFileName );
	#ifdef _OMP_
	printf( "ompMaxThreads:      %d\n", sia.ompMT );
	printf( "omp_schedule:       %s\n", omp_schedule );
	#endif
	printf( "FPT:                %s\n", fptMsg );
	printf( "sizeof(FPT):        %zu bytes\n", sizeof(FPT) );
	printf( "cudaBlockSize:      %d\n",  sia.cbs  );
	printf( "NUM_ITE:            %d\n", (UIN) NUM_ITE );
	printf( "CHUNK_SIZE:         %d\n", (UIN) CHUNK_SIZE ); fflush(stdout);
	return;
}


#ifndef HANDLE_CUDA_ERROR
	#define HANDLE_CUDA_ERROR( ceID ) { if ( ceID != hipSuccess ) { printf( "FILE: %s LINE: %d CUDA_ERROR: %s\n", __FILE__, __LINE__, hipGetErrorString( ceID ) ); fflush(stdout); printf( "\nvim %s +%d\n", __FILE__, __LINE__); exit( EXIT_FAILURE ); } }
#endif



static __host__ void getCudaDeviceCounter( int * counter )
{
	HANDLE_CUDA_ERROR( hipGetDeviceCount(counter) );
	HDL; printf( "cuda device properties\n" ); HDL;
	printf( "cudaDeviceCounter = %d\n\n", *counter );
	return;
}



static __host__ const char * getCudaComputeModeString( const int computeModeCode )
{
	switch( computeModeCode )
	{
		case 0: return "hipComputeModeDefault";
		case 1: return "hipComputeModeExclusive";
		case 2: return "hipComputeModeProhibited";
		case 3: return "hipComputeModeExclusiveProcess";
	}
	return "Unknown cudaComputeModeCode";
}



static __host__ void printCudaDeviceProperties( int cudaDeviceID )
{
	hipDeviceProp_t cudaDeviceProperties;
	HANDLE_CUDA_ERROR( hipGetDeviceProperties( &cudaDeviceProperties, cudaDeviceID ) );
	printf( "cudaDeviceID:                                     %d <-------------------\n", cudaDeviceID );
	printf( "cudaDeviceProperties.name:                        %s\n", cudaDeviceProperties.name );
	printf( "cudaDeviceProperties.totalGlobalMem:              %.1f %s\n", ( (float) cudaDeviceProperties.totalGlobalMem / (float) ( 1024 * 1024 * 1024 ) ), "GBytes" );
	printf( "cudaDeviceProperties.sharedMemPerBlock:           %.1f %s\n", ( (float) cudaDeviceProperties.sharedMemPerBlock / (float) 1024 ), "KBytes" );
	printf( "cudaDeviceProperties.textureAlignment:            %.1f %s\n", ( (float) cudaDeviceProperties.textureAlignment ), "Bytes" );
	printf( "cudaDeviceProperties.maxThreadsDim[0]:            %d\n", cudaDeviceProperties.maxThreadsDim[0] );
	printf( "cudaDeviceProperties.maxGridSize[0]:              %d\n", cudaDeviceProperties.maxGridSize[0] );
	printf( "cudaDeviceProperties.maxThreadsPerBlock:          %d\n", cudaDeviceProperties.maxThreadsPerBlock );
	printf( "cudaDeviceProperties.maxThreadsPerMultiProcessor: %d\n", cudaDeviceProperties.maxThreadsPerMultiProcessor );
	printf( "cudaDeviceProperties.multiProcessorCount:         %d\n", cudaDeviceProperties.multiProcessorCount  );
	printf( "cudaDeviceProperties.warpSize:                    %d\n", cudaDeviceProperties.warpSize );
	printf( "cudaDeviceProperties.canMapHostMemory:            %d\n", cudaDeviceProperties.canMapHostMemory );
	printf( "cudaDeviceProperties.major:                       %d\n", cudaDeviceProperties.major );
	printf( "cudaDeviceProperties.minor:                       %d\n", cudaDeviceProperties.minor );
	printf( "cudaDeviceProperties.regsPerBlock:                %d\n", cudaDeviceProperties.regsPerBlock );
	printf( "cudaDeviceProperties.multiProcessorCount:         %d\n", cudaDeviceProperties.multiProcessorCount );
	printf( "cudaDeviceProperties.computeMode:                 %s\n", getCudaComputeModeString( cudaDeviceProperties.computeMode ) );
	// set the bandwidth of the shared memory banks
	if ( sizeof(FPT) == 4 ) HANDLE_CUDA_ERROR( hipDeviceSetSharedMemConfig( hipSharedMemBankSizeFourByte  ) );
	if ( sizeof(FPT) == 8 ) HANDLE_CUDA_ERROR( hipDeviceSetSharedMemConfig( hipSharedMemBankSizeEightByte ) );
	// verify bandwidth of the shared memory banks
	hipSharedMemConfig csmc;
	HANDLE_CUDA_ERROR( hipDeviceGetSharedMemConfig( &csmc ) );
	unsigned short int bpb;
	if ( csmc == hipSharedMemBankSizeFourByte  ) bpb = 4;
	if ( csmc == hipSharedMemBankSizeEightByte ) bpb = 8;
	printf( "cudaDeviceSharedMemConfig:                        %1hu bytes\n", bpb );
	//HDL;
	return;
}


#ifndef TEST_POINTER
	#define TEST_POINTER( p ) { if ( p == NULL ) { fflush(stdout); printf( "\nFile: %s Line: %d Pointer: %s is null\n", __FILE__, __LINE__, #p ); fflush(stdout); exit( EXIT_FAILURE ); } }
#endif



typedef struct { UIN nrows; UIN nnz; UIN rmin; FPT rave; UIN rmax; FPT rsd; UIN bw; FPT * val; UIN * row; UIN * rowStart; UIN * rowEnd; UIN * col; UIN * rl; } str_matCSR;



static str_matCSR matrixReading( const char * matFileName )
{
	str_matCSR matCSR;
	if ( strstr( matFileName, ".csr" ) != NULL )
	{
		FILE * fh;
		fh = fopen( matFileName, "r" );
		if ( fh == NULL )
		{
			printf( "\nmatrixReading is unable to open .csr file\n\n" );
			exit( EXIT_FAILURE );
		}
		if ( fscanf( fh, "%d %d", &(matCSR.nrows), &(matCSR.nnz) ) != 2 ) ABORT;
		matCSR.val = (FPT *) malloc(   matCSR.nnz        * sizeof(FPT) ); TEST_POINTER( matCSR.val );
		matCSR.col = (UIN *) malloc(   matCSR.nnz        * sizeof(UIN) ); TEST_POINTER( matCSR.col );
		matCSR.row = (UIN *) malloc( ( matCSR.nrows + 1) * sizeof(UIN) ); TEST_POINTER( matCSR.row );
		matCSR.rl  = (UIN *) malloc(   matCSR.nrows      * sizeof(UIN) ); TEST_POINTER( matCSR.rl  );
		int i;
		for ( i = 0; i < ( matCSR.nnz ); i++ )
		{
			#if FP_TYPE == FPT_FLOAT
				if ( fscanf( fh, "%f %d\n",  &( matCSR.val[i] ), &( matCSR.col[i] ) ) != 2 ) ABORT;
			#else
				if ( fscanf( fh, "%lf %d\n", &( matCSR.val[i] ), &( matCSR.col[i] ) ) != 2 ) ABORT;
			#endif
		}
		for ( i = 0; i < ( matCSR.nrows + 1 ); i++ )
			if ( fscanf( fh, "%d", &(matCSR.row[i]) ) != 1 ) ABORT;
		fclose( fh );
	}
	else if ( strstr( matFileName, ".bin" ) != NULL )
	{
		size_t aux = 0;
		FILE * fh;
		fh = fopen( matFileName, "r" );
		if ( fh == NULL )
		{
			printf( "\nmatrixReading is unable to open .bin file\n\n" );
			exit( EXIT_FAILURE );
		}
		aux = fread( &(matCSR.nrows), sizeof(UIN), 1, fh );
		aux = fread( &(matCSR.nnz),   sizeof(UIN), 1, fh );
		matCSR.val = (FPT *) malloc(   matCSR.nnz        * sizeof(FPT) ); TEST_POINTER( matCSR.val );
		matCSR.col = (UIN *) malloc(   matCSR.nnz        * sizeof(UIN) ); TEST_POINTER( matCSR.col );
		matCSR.row = (UIN *) malloc( ( matCSR.nrows + 1) * sizeof(UIN) ); TEST_POINTER( matCSR.row );
		matCSR.rl  = (UIN *) malloc(   matCSR.nrows      * sizeof(UIN) ); TEST_POINTER( matCSR.rl  );
		aux = fread( matCSR.val, sizeof(FPT),   matCSR.nnz,         fh );
		aux = fread( matCSR.col, sizeof(UIN),   matCSR.nnz,         fh );
		aux = fread( matCSR.row, sizeof(UIN), ( matCSR.nrows + 1 ), fh );
		aux++;
		fclose(fh);
	}
	else
	{
		char buffer[128];
		strcpy( buffer, "matrixReading detected that " );
		strcat( buffer, matFileName );
		strcat( buffer, " has NOT .csr or .bin extension" );
		printf( "\n%s\n\n", buffer );
		exit( EXIT_FAILURE );
	}
	return( matCSR );
}



static void printMatrixStats( const char * matFileName, str_matCSR * matCSR )
{
	UIN    i, rl, rmin = 1e9, rmax = 0, j, bw = 0;
	int    dif;
	double rave1 = 0.0, rave2 = 0.0, rsd = 0.0;
	for ( i = 0; i < matCSR->nrows; i++ )
	{
		rl            = matCSR->row[i + 1] - matCSR->row[i];
		matCSR->rl[i] = rl;
		rave1         = rave1 +   rl;
		rave2         = rave2 + ( rl * rl );
		rmin          = (rmin<rl) ? rmin : rl;
		rmax          = (rmax>rl) ? rmax : rl;
		for ( j = matCSR->row[i]; j < matCSR->row[i+1]; j++ )
		{
			dif = abs( ((int) i) - ((int) matCSR->col[j]) );
			bw  = ( dif > bw ) ? dif : bw ;
		}
	}
	rave1 = rave1 / (double) (matCSR->nrows);
	rave2 = rave2 / (double) (matCSR->nrows);
	rsd   = sqrt( rave2 - ( rave1 * rave1 ) );
	matCSR->rmin = rmin;
	matCSR->rave = rave1;
	matCSR->rmax = rmax;
	matCSR->rsd  = rsd;
	matCSR->bw   = bw;
	char name[64];
	strcpy( name, matFileName );
	char * token1;
	const char deli[2] = ".";
	token1 = strtok( name, deli );
	strcat( token1, ".sta" );
	FILE * fh;
	fh = fopen( name, "w+" );
	fprintf( fh, "------------------------------------\n");
	fprintf( fh, "matrix's statistics\n");
	fprintf( fh, "------------------------------------\n");
	fprintf( fh, "name:  %28s\n",    matFileName );
	fprintf( fh, "nrows: %28d\n",    matCSR->nrows );
	fprintf( fh, "nnz:   %28d\n",    matCSR->nnz );
	fprintf( fh, "rmin:  %28d\n",    matCSR->rmin );
	fprintf( fh, "rave:  %28.2lf\n", matCSR->rave );
	fprintf( fh, "rmax:  %28d\n",    matCSR->rmax );
	fprintf( fh, "rsd:   %28.2lf\n", matCSR->rsd );
	fprintf( fh, "rsdp:  %28.2lf\n", ( ( rsd / rave1 ) * 100 ) );
	fprintf( fh, "bw:    %28d\n",    matCSR->bw );
	fclose( fh );
	return;
}



typedef struct { char name[48]; double mfp; double beta; double ct; } str_formatData;



#ifndef GT
	#define GT( t ) { clock_gettime( CLOCK_MONOTONIC, &t ); }
#endif



static double measure_time( const struct timespec t2, const struct timespec t1 )
{
	double t = (double) ( t2.tv_sec - t1.tv_sec ) + ( (double) ( t2.tv_nsec - t1.tv_nsec ) ) * 1e-9;
	return( t );
}



static str_formatData getFormatDataCSR( str_matCSR * matCSR )
{
	// define local variables
	UIN i, ii;
	double ti = 0.0, tt = 0.0;
	struct timespec t1, t2;
	matCSR->rowStart = (UIN *) calloc( matCSR->nrows, sizeof(UIN) ); TEST_POINTER( matCSR->rowStart );
	matCSR->rowEnd   = (UIN *) calloc( matCSR->nrows, sizeof(UIN) ); TEST_POINTER( matCSR->rowEnd   );
	for ( i = 0; i < NUM_ITE; i++ )
	{
		GT( t1 );
		for ( ii = 0; ii < matCSR->nrows; ii++ )
		{
			matCSR->rowStart[ii] = matCSR->row[ii];
			matCSR->rowEnd[ii]   = matCSR->row[ii+1];
		}
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	// format's name
	str_formatData fd;
	strcpy( fd.name, "fcsr" );
	// CSR memory footprint
	fd.mfp =          (double) (   matCSR->nnz         * sizeof(FPT) ); // val
	fd.mfp = fd.mfp + (double) (   matCSR->nnz         * sizeof(UIN) ); // col
	fd.mfp = fd.mfp + (double) ( ( matCSR->nrows + 1 ) * sizeof(UIN) ); // row
	fd.mfp = fd.mfp + (double) (   matCSR->nrows       * sizeof(FPT) ); // vec
	// CSR occupancy ( beta )
	fd.beta = ( (double) matCSR->nnz / (double) matCSR->nnz );
	// CSR conversion time (conversion time for MKL functions)
	fd.ct = tt / (double) NUM_ITE;
	return( fd );
}



static void init_vec( const UIN ompNT, const UIN len, FPT * vec )
{
	UIN i;
	#pragma omp parallel for default(shared) private(i) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( i = 0 ; i < len; i++ )
		vec[i] = (FPT) i;
	return;
}



static void fill_array( const UIN ompNT, const UIN len, FPT val, FPT * array )
{
	UIN i;
	#pragma omp parallel for default(shared) private(i) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( i = 0; i < len; i++ )
		array[i] = val;
	return;
}



static void ncsr( const UIN ompNT, const str_matCSR matCSR, const FPT * vec, FPT * res )
{
	UIN i, j;
	FPT aux;
	#pragma omp parallel for default(shared) private(i,j,aux) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( i = 0; i < matCSR.nrows; i++ )
	{
		aux = (FPT) 0;
		for ( j = matCSR.row[i]; j < matCSR.row[i+1]; j++ )
		{
			aux = aux + matCSR.val[j] * vec[matCSR.col[j]];
		}
		res[i] = aux;
	}
	return;
}



typedef struct { double aErr; double rErr; UIN pos; } str_err;



static void get_errors( const UIN len, const FPT * ar, const FPT * ac, str_err * sErr )
{
	double dif, maxDif = 0.0;
	double val, maxVal = 0.0;
	UIN pos = 0;
	UIN i;
	for ( i = 0; i < len; i++ )
	{
		val = fabs(ar[i]);
		if ( val > maxVal ) maxVal = val;
		dif = fabs( fabs(ac[i]) - val );
		if ( dif > maxDif )
		{
			maxDif = dif;
			pos    = i;
		}
	}
	sErr->aErr = maxDif;
	sErr->rErr = maxDif/maxVal;
	sErr->pos  = pos;
	return;
}



typedef struct { char name[48]; double et; double ot; double flops; str_err sErr; } str_res;



static str_res test_ncsr( const UIN ompNT, const str_matCSR matCSR, const FPT * vec, const FPT * ref )
{
	// timed iterations
	double ti = 0.0, tt = 0.0;
	struct timespec t1, t2;
	FPT * res = (FPT *) calloc( matCSR.nrows, sizeof(FPT) ); TEST_POINTER( res );
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		GT( t1 );
		ncsr( ompNT, matCSR, vec, res );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	// store results
	str_res sr;
	strcpy( sr.name, "ncsr" );
	sr.et    = tt / (double) NUM_ITE;
	sr.ot    = 0.0;
	sr.flops = ( 2.0 * ( (double) matCSR.nnz ) ) / sr.et;
	get_errors( matCSR.nrows, ref, res, &(sr.sErr) );
	free( res );
	return( sr );
}



static __global__ void gcsr( const UIN nrows, const FPT * val, const UIN * col, const UIN * row, const FPT * x, FPT * y )
{
	const UIN rowID = blockIdx.x * blockDim.x + threadIdx.x;
	if ( rowID < nrows )
	{
		UIN i;
		FPT aux = 0.0;
		for ( i = row[rowID]; i < row[rowID + 1]; i++ )
			aux = aux + val[i] * x[col[i]];
		y[rowID] = aux;
	}
	return;
}



static __host__ str_res test_gcsr( const UIN cudaBlockSize, const str_matCSR matCSR, const FPT * vec, const FPT * ref )
{
	// get parameters
	const UIN        nrows = matCSR.nrows;
	const UIN          nnz = matCSR.nnz;
	const UIN cudaBlockNum = ( nrows + cudaBlockSize - 1 ) / cudaBlockSize;
	// allocate memory on GPU
	FPT * d_val; HANDLE_CUDA_ERROR( hipMalloc( &d_val,          nnz * sizeof(FPT) ) ); TEST_POINTER( d_val );
	UIN * d_col; HANDLE_CUDA_ERROR( hipMalloc( &d_col,          nnz * sizeof(UIN) ) ); TEST_POINTER( d_col );
	UIN * d_row; HANDLE_CUDA_ERROR( hipMalloc( &d_row, (nrows + 1 ) * sizeof(UIN) ) ); TEST_POINTER( d_row );
	FPT * d_vec; HANDLE_CUDA_ERROR( hipMalloc( &d_vec,        nrows * sizeof(FPT) ) ); TEST_POINTER( d_vec );
	FPT * d_res; HANDLE_CUDA_ERROR( hipMalloc( &d_res,        nrows * sizeof(FPT) ) ); TEST_POINTER( d_res );
	// copy necessary arrays to device
	HANDLE_CUDA_ERROR( hipMemcpy( d_val, matCSR.val,          nnz * sizeof(FPT), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_col, matCSR.col,          nnz * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_row, matCSR.row, ( nrows + 1 )* sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_vec, vec,               nrows * sizeof(FPT), hipMemcpyHostToDevice ) );
	// create events for time measuring
	hipEvent_t cet1; HANDLE_CUDA_ERROR( hipEventCreate( &cet1 ) );
	hipEvent_t cet2; HANDLE_CUDA_ERROR( hipEventCreate( &cet2 ) );
	// timed iterations
	float ti = 0.0f, tt = 0.0f;
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		HANDLE_CUDA_ERROR( hipEventRecord( cet1 ) );
		gcsr <<<cudaBlockNum, cudaBlockSize>>> ( nrows, d_val, d_col, d_row, d_vec, d_res );
		HANDLE_CUDA_ERROR( hipEventRecord( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventSynchronize( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventElapsedTime( &ti, cet1, cet2 ) );
		tt = tt + ti;
	}
	// destroy events for time measuring
	HANDLE_CUDA_ERROR( hipEventDestroy( cet1 ) );
	HANDLE_CUDA_ERROR( hipEventDestroy( cet2 ) );
	// copy result from device
	FPT * res = (FPT *) calloc( matCSR.nrows, sizeof(FPT) ); TEST_POINTER( res );
	HANDLE_CUDA_ERROR( hipMemcpy( res, d_res, nrows * sizeof(FPT), hipMemcpyDeviceToHost ) );
	// free device memory
	HANDLE_CUDA_ERROR( hipFree( d_val ) );
	HANDLE_CUDA_ERROR( hipFree( d_col ) );
	HANDLE_CUDA_ERROR( hipFree( d_row ) );
	HANDLE_CUDA_ERROR( hipFree( d_vec ) );
	HANDLE_CUDA_ERROR( hipFree( d_res ) );
	// store results
	str_res sr;
	strcpy( sr.name, "gcsr" );
	sr.et    = ( (double) tt / (double) NUM_ITE ) * 1e-3;
	sr.flops = ( 2.0 * ( (double) matCSR.nnz ) ) / sr.et;
	get_errors( matCSR.nrows, ref, res, &(sr.sErr) );
	// free cpu memory
	free( res );
	return( sr );
}



static __host__ const char * cusparseGetErrorMessage( hipsparseStatus_t statusID )
{
	switch(statusID)
	{
		case HIPSPARSE_STATUS_NOT_INITIALIZED:           return "HIPSPARSE_STATUS_NOT_INITIALIZED";
		case HIPSPARSE_STATUS_ALLOC_FAILED:              return "HIPSPARSE_STATUS_ALLOC_FAILED";
		case HIPSPARSE_STATUS_INVALID_VALUE:             return "HIPSPARSE_STATUS_INVALID_VALUE";
		case HIPSPARSE_STATUS_ARCH_MISMATCH:             return "HIPSPARSE_STATUS_ARCH_MISMATCH";
		case HIPSPARSE_STATUS_MAPPING_ERROR:             return "HIPSPARSE_STATUS_MAPPING_ERROR";
		case HIPSPARSE_STATUS_EXECUTION_FAILED:          return "HIPSPARSE_STATUS_EXECUTION_FAILED";
		case HIPSPARSE_STATUS_INTERNAL_ERROR:            return "HIPSPARSE_STATUS_INTERNAL_ERROR";
		case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
	}
	return "<cusparse unknown>";
}



#ifndef HANDLE_CUSPARSE_ERROR
	#define HANDLE_CUSPARSE_ERROR( cseID ) { if ( cseID != HIPSPARSE_STATUS_SUCCESS ) { printf( "FILE: %s LINE: %d CUBLAS_ERROR: %s\n", __FILE__, __LINE__, cusparseGetErrorMessage( cseID ) ); printf( "\nvim %s +%d\n", __FILE__, __LINE__); exit( EXIT_FAILURE ); } }
#endif



static __host__ str_res test_gcucsr( const str_matCSR matCSR, const FPT * vec, const FPT * ref )
{
	// get parameteres for cuSPARSE
	const UIN                     nrows = matCSR.nrows;
	const UIN                       nnz = matCSR.nnz;
	      hipsparseHandle_t    cusparseH = NULL;
	const cusparseAlgMode_t  cusparseAM = CUSPARSE_ALG1;
	const hipsparseOperation_t cusparseO = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	      hipsparseMatDescr_t cusparseMD = NULL;
	      size_t    cudaSpaceBufferSize;
	const FPT                      zero = (FPT)  0;
	const FPT                       one = (FPT)  1;
	#if FP_TYPE == FP_FLOAT
		hipDataType cudaDT = HIP_R_32F;
	#else
		hipDataType cudaDT = HIP_R_64F;
	#endif
	// create handlers for cuSPARSE
	HANDLE_CUSPARSE_ERROR( hipsparseCreate(&cusparseH) );
	HANDLE_CUSPARSE_ERROR( hipsparseCreateMatDescr( &cusparseMD ) );
	HANDLE_CUSPARSE_ERROR( hipsparseSetMatIndexBase( cusparseMD, HIPSPARSE_INDEX_BASE_ZERO ) );
	HANDLE_CUSPARSE_ERROR( hipsparseSetMatType( cusparseMD, HIPSPARSE_MATRIX_TYPE_GENERAL ) );
	// allocate memory on GPU
	FPT * d_val; HANDLE_CUDA_ERROR( hipMalloc( &d_val,           nnz * sizeof(FPT) ) ); TEST_POINTER( d_val );
	int * d_col; HANDLE_CUDA_ERROR( hipMalloc( &d_col,           nnz * sizeof(int) ) ); TEST_POINTER( d_col );
	int * d_row; HANDLE_CUDA_ERROR( hipMalloc( &d_row, ( nrows + 1 ) * sizeof(int) ) ); TEST_POINTER( d_row );
	FPT * d_vec; HANDLE_CUDA_ERROR( hipMalloc( &d_vec,         nrows * sizeof(FPT) ) ); TEST_POINTER( d_vec );
	FPT * d_res; HANDLE_CUDA_ERROR( hipMalloc( &d_res,         nrows * sizeof(FPT) ) ); TEST_POINTER( d_res );
	// copy necessary arrays to device
	HANDLE_CUDA_ERROR( hipMemcpy( d_val, matCSR.val,           nnz * sizeof(FPT), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_col, matCSR.col,           nnz * sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_row, matCSR.row, ( nrows + 1 ) * sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_vec, vec,                nrows * sizeof(FPT), hipMemcpyHostToDevice ) );
	// get space buffer for cusparseCsrmvEx
	HANDLE_CUSPARSE_ERROR( cusparseCsrmvEx_bufferSize( cusparseH, cusparseAM, cusparseO, matCSR.nrows, matCSR.nrows, matCSR.nnz, &one, cudaDT, cusparseMD, \
                                                        d_val, cudaDT, d_row, d_col, d_vec, cudaDT, &zero, cudaDT, d_res, cudaDT, cudaDT, &cudaSpaceBufferSize ) );
	void * cudaSpaceBuffer; HANDLE_CUDA_ERROR( hipMalloc( &cudaSpaceBuffer, cudaSpaceBufferSize ) );
	// create events for time measuring
	hipEvent_t cet1; HANDLE_CUDA_ERROR( hipEventCreate( &cet1 ) );
	hipEvent_t cet2; HANDLE_CUDA_ERROR( hipEventCreate( &cet2 ) );
	// timed iterations
	float ti = 0.0, tt = 0.0;
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		HANDLE_CUDA_ERROR( hipEventRecord( cet1 ) );
		HANDLE_CUSPARSE_ERROR( cusparseCsrmvEx( cusparseH, cusparseAM, cusparseO, matCSR.nrows, matCSR.nrows, matCSR.nnz, &one, cudaDT, cusparseMD, \
                                                  d_val, cudaDT, d_row, d_col, d_vec, cudaDT, &zero, cudaDT, d_res, cudaDT, cudaDT, cudaSpaceBuffer ) );
		HANDLE_CUDA_ERROR( hipEventRecord( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventSynchronize( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventElapsedTime( &ti, cet1, cet2 ) );
		tt = tt + ti;
	}
	// destroy events for time measuring
	HANDLE_CUDA_ERROR( hipEventDestroy( cet1 ) );
	HANDLE_CUDA_ERROR( hipEventDestroy( cet2 ) );
	// copy result from device
	FPT * res = (FPT *) calloc( nrows, sizeof(FPT) ); TEST_POINTER( res );
	HANDLE_CUDA_ERROR( hipMemcpy( res, d_res, nrows * sizeof(FPT), hipMemcpyDeviceToHost ) );
	// free device memory
	HANDLE_CUDA_ERROR( hipFree( d_val ) );
	HANDLE_CUDA_ERROR( hipFree( d_col ) );
	HANDLE_CUDA_ERROR( hipFree( d_row ) );
	HANDLE_CUDA_ERROR( hipFree( d_vec ) );
	HANDLE_CUDA_ERROR( hipFree( d_res ) );
	// store results
	str_res sr;
	strcpy( sr.name, "gcucsr" );
	sr.et    = ( (double) tt / (double) NUM_ITE ) * 1e-3;
	sr.flops = ( (double) matCSR.nnz * 2.0 ) / sr.et;
	get_errors( matCSR.nrows, ref, res, &(sr.sErr) );
	// free cpu memory
	free( res );
	return( sr );
}



typedef struct { UIN ind; UIN val; } str_pair;



typedef struct { UIN nrows; UIN nnz; UIN chunkNum; UIN lenVC; UIN * permi; UIN * nmc; UIN * chp; FPT * val; UIN * col; } str_matK1;



static int orderFunction( const void * ele1, const void * ele2 )
{
	return (  ( (str_pair *) ele2 )->val - ( (str_pair *) ele1 )->val  );
}



static void getArrayPermiK1( const str_matCSR matCSR, str_matK1 * matK1 )
{
	str_pair * list = (str_pair *) malloc( matCSR.nrows * sizeof(str_pair) ); TEST_POINTER( list );
	UIN i;
	for ( i = 0; i < matK1->nrows; i++ )
	{
		list[i].ind = i;
		list[i].val = matCSR.rl[i];
	}
	qsort( list, matK1->nrows, sizeof(str_pair), orderFunction );
	for ( i = 0; i < matK1->nrows; i++ )
		matK1->permi[i] = list[i].ind;
	free( list );
	return;
}



static UIN getArraysNmcChpK1( const str_matCSR matCSR, str_matK1 * matK1 )
{
	UIN i, p, n, l = 0, chunkNum = ( matCSR.nrows + CHUNK_SIZE - 1 ) / CHUNK_SIZE;
	for ( i = 0 ; i < chunkNum; i++ )
	{
		p             = matK1->permi[i * CHUNK_SIZE];
		n             = matCSR.rl[p];
		matK1->nmc[i] = n;
		l             = l + CHUNK_SIZE * n;
	}
	for ( i = 1; i < matK1->chunkNum; i++ )
		matK1->chp[i] = matK1->chp[i-1] + ( matK1->nmc[i-1] * CHUNK_SIZE );
	return l;
}



static void getArraysValColK1( const str_matCSR matCSR, str_matK1 * matK1 )
{
	const UIN chunkNum = matK1->chunkNum;
	UIN chunkID, rid, row, posCSR, rowOff, posK1;
	for ( chunkID = 0; chunkID < chunkNum; chunkID++ )
	{
		for ( rid = 0; rid < CHUNK_SIZE; rid++ )
		{
			row = chunkID * CHUNK_SIZE + rid;
			if ( row == matCSR.nrows ) return;
			row = matK1->permi[row];
			for ( posCSR = matCSR.row[row], rowOff = 0; posCSR < matCSR.row[row + 1]; posCSR++, rowOff++ )
			{
				posK1             = matK1->chp[chunkID] + rowOff * CHUNK_SIZE + rid;
				matK1->val[posK1] = matCSR.val[posCSR];
				matK1->col[posK1] = matCSR.col[posCSR];
			}
		}
	}
	return;
}



static str_formatData getFormatDataK1( const UIN blockSize, const str_matCSR matCSR, const FPT * vec, str_matK1 * matK1 )
{
	// get K1 parameters
	matK1->nrows     = matCSR.nrows;
	matK1->nnz       = matCSR.nnz;
	matK1->chunkNum  = ( matCSR.nrows + CHUNK_SIZE - 1 ) / CHUNK_SIZE;
	matK1->permi     = (UIN *) calloc( ( matK1->chunkNum + 1 ) * CHUNK_SIZE, sizeof(UIN) ); TEST_POINTER( matK1->permi );
	matK1->nmc       = (UIN *) calloc(   matK1->chunkNum,                    sizeof(UIN) ); TEST_POINTER( matK1->nmc   );
	matK1->chp       = (UIN *) calloc(   matK1->chunkNum,                    sizeof(UIN) ); TEST_POINTER( matK1->chp   );
	UIN i;
	for ( i = 0; i < ( matK1->chunkNum + 1 ) * CHUNK_SIZE; i++ )
		matK1->permi[i] = 0;
	// get matK1
	struct timespec t1, t2;
	double ti = 0.0, tt = 0.0, tc = 0.0;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		GT( t1 );
		getArrayPermiK1( matCSR, matK1 );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	ti = tt / (double) NUM_ITE;
	tc = tc + ti;
	tt = 0.0;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		GT( t1 );
		matK1->lenVC = getArraysNmcChpK1( matCSR, matK1 );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	ti = tt / (double) NUM_ITE;
	tc = tc + ti;
	matK1->val = (FPT *) calloc( matK1->lenVC, sizeof(FPT) ); TEST_POINTER( matK1->val );
	matK1->col = (UIN *) calloc( matK1->lenVC, sizeof(UIN) ); TEST_POINTER( matK1->col );
	tt = 0.0;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		GT( t1 );
		getArraysValColK1( matCSR, matK1 );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	ti = tt / (double) NUM_ITE;
	tc = tc + ti;
	// format's name
	str_formatData fd;
	strcpy( fd.name, "fk1" );
	// K1 memory footprint
	fd.mfp =          (double) ( matK1->chunkNum * sizeof(UIN) ); // nmc
	fd.mfp = fd.mfp + (double) ( matK1->chunkNum * sizeof(UIN) ); // chp
	fd.mfp = fd.mfp + (double) ( matK1->lenVC    * sizeof(FPT) ); // val
	fd.mfp = fd.mfp + (double) ( matK1->lenVC    * sizeof(UIN) ); // col
	fd.mfp = fd.mfp + (double) ( matK1->nrows    * sizeof(UIN) ); // permi
	fd.mfp = fd.mfp + (double) ( matK1->nrows    * sizeof(FPT) ); // vec
	// K1 occupancy ( beta )
	fd.beta = ( (double) matK1->nnz / (double) (matK1->lenVC) );
	// K1 conversion time
	fd.ct = tc;
	return( fd );
}




static __global__ void gk1( const int NROWS, const FPT * val, const UIN * col, const UIN * nmc, const UIN * chp, const UIN * permi, const FPT * x, FPT * y )
{
	const UIN gid = blockIdx.x * blockDim.x + threadIdx.x;
	const UIN lid = threadIdx.x;
	const UIN cid = gid / CHUNK_SIZE;
	const UIN wid = lid & ( CHUNK_SIZE - 1 );
	if ( gid < NROWS )
	{
		UIN to = chp[cid] + wid;
		UIN ul = nmc[cid] * CHUNK_SIZE + to;
		FPT sum = val[to] * x[col[to]];
		for ( to = ( to + CHUNK_SIZE ); to < ul; to = ( to + CHUNK_SIZE ) )
			sum = sum + val[to] * x[col[to]];
		y[permi[gid]] = sum;
	}
	return;
}



static __host__ str_res test_gk1( const UIN cudaBlockSize, const str_matK1 matK1, const FPT * vec, const FPT * ref )
{
	// 
	UIN cudaBlockNum = ( matK1.nrows + cudaBlockSize - 1 ) / cudaBlockSize;
	// allocate memory on GPU
	FPT * d_val;   HANDLE_CUDA_ERROR( hipMalloc( &d_val,   matK1.lenVC                 * sizeof(FPT) ) ); TEST_POINTER( d_val   );
	UIN * d_col;   HANDLE_CUDA_ERROR( hipMalloc( &d_col,   matK1.lenVC                 * sizeof(UIN) ) ); TEST_POINTER( d_col   );
	UIN * d_nmc;   HANDLE_CUDA_ERROR( hipMalloc( &d_nmc,   matK1.chunkNum              * sizeof(UIN) ) ); TEST_POINTER( d_nmc   );
	UIN * d_chp;   HANDLE_CUDA_ERROR( hipMalloc( &d_chp,   matK1.chunkNum              * sizeof(UIN) ) ); TEST_POINTER( d_chp   );
	UIN * d_permi; HANDLE_CUDA_ERROR( hipMalloc( &d_permi, matK1.chunkNum * CHUNK_SIZE * sizeof(UIN) ) ); TEST_POINTER( d_permi );
	FPT * d_vec;   HANDLE_CUDA_ERROR( hipMalloc( &d_vec,   matK1.nrows                 * sizeof(FPT) ) ); TEST_POINTER( d_vec   );
	FPT * d_res;   HANDLE_CUDA_ERROR( hipMalloc( &d_res,   matK1.nrows                 * sizeof(FPT) ) ); TEST_POINTER( d_res   );
	// copy necessary arrays to device
	HANDLE_CUDA_ERROR( hipMemcpy( d_val,   matK1.val,   matK1.lenVC                 * sizeof(FPT), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_col,   matK1.col,   matK1.lenVC                 * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_nmc,   matK1.nmc,   matK1.chunkNum              * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_chp,   matK1.chp,   matK1.chunkNum              * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_permi, matK1.permi, matK1.chunkNum * CHUNK_SIZE * sizeof(UIN), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_vec,   vec,         matK1.nrows                 * sizeof(FPT), hipMemcpyHostToDevice ) );
	// create events for time measuring
	hipEvent_t cet1; HANDLE_CUDA_ERROR( hipEventCreate( &cet1 ) );
	hipEvent_t cet2; HANDLE_CUDA_ERROR( hipEventCreate( &cet2 ) );
	// timed iterations
	float ti = 0.0f, tt = 0.0f;
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		HANDLE_CUDA_ERROR( hipEventRecord( cet1 ) );
		gk1 <<<cudaBlockNum, cudaBlockSize>>> (  matK1.nrows, d_val, d_col, d_nmc, d_chp, d_permi, d_vec, d_res );
		HANDLE_CUDA_ERROR( hipEventRecord( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventSynchronize( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventElapsedTime( &ti, cet1, cet2 ) );
		tt = tt + ti;
	}
	// destroy events for time measuring
	HANDLE_CUDA_ERROR( hipEventDestroy( cet1 ) );
	HANDLE_CUDA_ERROR( hipEventDestroy( cet2 ) );
	// copy result from device
	FPT * res = (FPT *) malloc( matK1.nrows * sizeof(FPT) ); TEST_POINTER( res );
	HANDLE_CUDA_ERROR( hipMemcpy( res, d_res, matK1.nrows * sizeof(FPT), hipMemcpyDeviceToHost ) );
	// free device memory
	HANDLE_CUDA_ERROR( hipFree( d_val   ) );
	HANDLE_CUDA_ERROR( hipFree( d_col   ) );
	HANDLE_CUDA_ERROR( hipFree( d_nmc   ) );
	HANDLE_CUDA_ERROR( hipFree( d_chp   ) );
	HANDLE_CUDA_ERROR( hipFree( d_permi ) );
	HANDLE_CUDA_ERROR( hipFree( d_vec   ) );
	HANDLE_CUDA_ERROR( hipFree( d_res   ) );
	// store results
	str_res sr;
	strcpy( sr.name, "gk1" );
	sr.et    = ( (double) tt / (double) NUM_ITE ) * 1e-3;
	sr.flops = ( 2.0 * ( (double) matK1.nnz ) ) / sr.et;
	get_errors( matK1.nrows, ref, res, &(sr.sErr) );
	// free cpu memory
	free( res );
	return( sr );
}



typedef struct { UIN nrows; UIN nnz; UIN hbs; UIN lenAX; UIN lenBRP; UIN lenMAPX; FPT * ax; UIN * brp; UIN * mapx; } str_matAXC;



static UIN get_brpAXC( const str_matCSR matCSR, str_matAXC * matAXC )
{
	const UIN hbs   = matAXC->hbs;
	const UIN nrows = matAXC->nrows;
	      UIN rowID, brickNum;
	for ( rowID = 0; rowID < nrows; rowID++ )
	{
		brickNum               = ( matCSR.rl[rowID] + hbs - 1 ) / hbs;
		matAXC->brp[rowID + 1] = matAXC->brp[rowID]  + ( 2 * brickNum * hbs );
	}
	return( matAXC->brp[matAXC->nrows] );
}



static void get_axAXC( const UIN ompNT, const str_matCSR matCSR, const FPT * vec, str_matAXC * matAXC )
{
	const UIN hbs   = matAXC->hbs;
	const UIN nrows = matAXC->nrows;
	      UIN rowID, posAX, counter, posCSR;
	#pragma omp parallel for default(shared) private(rowID,posAX,counter,posCSR) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( rowID = 0; rowID < nrows; rowID++ )
	{
		posAX   = matAXC->brp[rowID];
		counter = 0;
		for ( posCSR = matCSR.row[rowID]; posCSR < matCSR.row[rowID + 1]; posCSR++ )
		{
			matAXC->ax[posAX]       = matCSR.val[posCSR];
			matAXC->ax[posAX + hbs] = vec[matCSR.col[posCSR]];
			if ( counter == (hbs - 1) )
			{
				posAX  = posAX + 1 + hbs;
				counter = 0;
			}
			else
			{
				posAX++;
				counter++;
			}
		}
	}
	return;
}



static void get_mapxAXC( const UIN ompNT, const str_matCSR matCSR, str_matAXC * matAXC )
{
	const UIN nrows = matAXC->nrows;
	      UIN rowID, pos1, pos2, pos, eleID;
	#pragma omp parallel for default(shared) private(rowID,pos1,pos2,pos,eleID) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( rowID = 0; rowID < nrows; rowID++ )
	{
		pos1 = matCSR.row[rowID];
		pos2 = matCSR.row[rowID+1];
		pos  = matAXC->brp[rowID]>>1;
		for ( eleID = pos1; eleID < pos2; eleID++ )
		{
			matAXC->mapx[pos] = matCSR.col[eleID];
			pos++;
		}
	}
	return;
}



static str_formatData getFormatDataAXC( const UIN ompNT, const UIN hbs, const str_matCSR matCSR, const FPT * vec, str_matAXC * matAXC )
{
	// get AXC parameters
	matAXC->nrows  = matCSR.nrows;
	matAXC->nnz    = matCSR.nnz;
	matAXC->hbs    = hbs;
	matAXC->lenBRP = matCSR.nrows + 1;
	matAXC->brp    = (UIN *) calloc( matAXC->lenBRP, sizeof(UIN) ); TEST_POINTER( matAXC->brp  );
	// get matAXC
	struct timespec t1, t2;
	double ti = 0.0, tt = 0.0, tc = 0.0;
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		GT( t1 );
		matAXC->lenAX = get_brpAXC( matCSR, matAXC );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	ti = tt / (double) NUM_ITE;
	tc = tc + ti;
	matAXC->ax      = (FPT *) calloc( matAXC->lenAX,   sizeof(FPT) ); TEST_POINTER( matAXC->ax );
	matAXC->lenMAPX = (matAXC->lenAX >> 1) + 8;
	matAXC->mapx    = (UIN *) calloc( matAXC->lenMAPX, sizeof(UIN) ); TEST_POINTER( matAXC-> mapx );
	tt = 0.0;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		GT( t1 );
		get_mapxAXC( ompNT, matCSR, matAXC );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	ti = tt / (double) NUM_ITE;
	tc = tc + ti;
	tt = 0.0;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		GT( t1 );
		get_axAXC( ompNT, matCSR, vec, matAXC );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	ti = tt / (double) NUM_ITE;
	tc = tc + ti;
	// format's name
	str_formatData fd;
	strcpy( fd.name, "faxc" );
	// AXC memory footprint
	fd.mfp =          (double) ( matAXC->lenAX  * sizeof(FPT) ); // ax
	fd.mfp = fd.mfp + (double) ( matAXC->lenBRP * sizeof(UIN) ); // brp ( stores the starting address of a row )
	// AXC occupancy ( beta )
	fd.beta = ( (double) matAXC->nnz / (double) (matAXC->lenAX >> 1) );
	// AXC conversion time
	fd.ct = tc;
	return( fd );
}



#ifndef FULL_MASK
	#define FULL_MASK 0xffffffff
#endif



static __global__ void gaxc( const FPT * ax, const UIN * brp, FPT * y )
{
	const UIN tidGRID = blockIdx.x * blockDim.x + threadIdx.x;
	const UIN widGRID = tidGRID >> 5;
	const UIN tidWARP = tidGRID & 31;
	const UIN p1      = brp[widGRID]   + tidWARP;
	const UIN p2      = brp[widGRID+1] + tidWARP;
	      UIN pAX;
	      FPT val = 0.0, red = 0.0;
	for ( pAX = p1; pAX < p2; pAX = pAX + 64 )
	{
		val = ax[pAX] * ax[pAX+32];
		val = val + __shfl_down_sync( FULL_MASK, val, 16 );
		val = val + __shfl_down_sync( FULL_MASK, val,  8 );
		val = val + __shfl_down_sync( FULL_MASK, val,  4 );
		val = val + __shfl_down_sync( FULL_MASK, val,  2 );
		val = val + __shfl_down_sync( FULL_MASK, val,  1 );
		red = red + val;
	}
	if (tidWARP == 0) y[widGRID] = red;
	return;
}



static __host__ str_res test_gaxc( const UIN cudaBlockSize, const str_matAXC matAXC, const FPT * ref )
{
	// 
	UIN cudaBlockNum = matAXC.nrows * 32;
	    cudaBlockNum = ( cudaBlockNum + cudaBlockSize - 1 ) / cudaBlockSize;
	// allocate memory on GPU
	FPT * d_ax;    HANDLE_CUDA_ERROR( hipMalloc( &d_ax,    matAXC.lenAX                * sizeof(FPT) ) ); TEST_POINTER( d_ax    );
	UIN * d_brp;   HANDLE_CUDA_ERROR( hipMalloc( &d_brp,   matAXC.lenBRP               * sizeof(UIN) ) ); TEST_POINTER( d_brp   );
	FPT * d_res;   HANDLE_CUDA_ERROR( hipMalloc( &d_res,   matAXC.nrows                * sizeof(FPT) ) ); TEST_POINTER( d_res   );
	// copy necessary arrays to device
	HANDLE_CUDA_ERROR( hipMemcpy( d_ax,    matAXC.ax,    matAXC.lenAX                  * sizeof(FPT), hipMemcpyHostToDevice ) );
	HANDLE_CUDA_ERROR( hipMemcpy( d_brp,   matAXC.brp,   matAXC.lenBRP                 * sizeof(UIN), hipMemcpyHostToDevice ) );
	// create events for time measuring
	hipEvent_t cet1; HANDLE_CUDA_ERROR( hipEventCreate( &cet1 ) );
	hipEvent_t cet2; HANDLE_CUDA_ERROR( hipEventCreate( &cet2 ) );
	// timed iterations
	float ti = 0.0f, tt = 0.0f;
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		HANDLE_CUDA_ERROR( hipEventRecord( cet1 ) );
		gaxc <<<cudaBlockNum, cudaBlockSize>>> ( d_ax, d_brp, d_res );
		HANDLE_CUDA_ERROR( hipEventRecord( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventSynchronize( cet2 ) );
		HANDLE_CUDA_ERROR( hipEventElapsedTime( &ti, cet1, cet2 ) );
		tt = tt + ti;
	}
	// destroy events for time measuring
	HANDLE_CUDA_ERROR( hipEventDestroy( cet1 ) );
	HANDLE_CUDA_ERROR( hipEventDestroy( cet2 ) );
	// copy result from device
	FPT * res = (FPT *) malloc( matAXC.nrows * sizeof(FPT) ); TEST_POINTER( res );
	HANDLE_CUDA_ERROR( hipMemcpy( res, d_res, matAXC.nrows * sizeof(FPT), hipMemcpyDeviceToHost ) );
	// free device memory
	HANDLE_CUDA_ERROR( hipFree( d_ax    ) );
	HANDLE_CUDA_ERROR( hipFree( d_brp   ) );
	HANDLE_CUDA_ERROR( hipFree( d_res   ) );
	// store results
	str_res sr;
	strcpy( sr.name, "gaxc" );
	sr.et    = ( (double) tt / (double) NUM_ITE ) * 1e-3;
	sr.flops = ( 2.0 * ( (double) matAXC.nnz ) ) / sr.et;
	get_errors( matAXC.nrows, ref, res, &(sr.sErr) );
	// free cpu memory
	free( res );
	return( sr );
}



/*
typedef struct{ UIN nrows; UIN nnz; char mode[8]; UIN tileHW; UIN tileH; UIN logTH; UIN tileN; UIN lenAX; UIN lenSEC; UIN lenCON; UIN log; UIN bs; FPT * ax; UIN * sec; UIN * con; } str_matAXT;



static void getArraysLenAXT( const str_matCSR matCSR, str_matAXT * matAXT )
{
	const UIN  nrows = matAXT->nrows;
	const UIN    thw = matAXT->tileHW;
	const UIN     th = matAXT->tileH;
	const UIN    ths = thw * th;
	const UIN grpLen = (th == 1) ? (thw) : (th) ;
	char mode[8];
	strcpy( mode, matAXT->mode );
	      UIN rowID = 0, rowStartPos = 0, rowOffT, rowOffR, rowOffC, pos, rowLen, positions, columns, totalColumns = 0, totalTiles;
	for ( ; rowID < nrows; rowID++ )
	{
		           rowOffT = ( (rowStartPos + ths)/ths ) - 1;
		           rowOffR =    rowStartPos % th;
		           rowOffC = ( (rowStartPos + th)/th ) - 1 - (rowOffT * thw);
		               pos = rowOffT * (2 * ths) + rowOffR * (2 * thw) + rowOffC;
		matAXT->con[rowID] = pos;
		            rowLen = matCSR.rl[rowID];
		         positions = ( strcmp( mode, "UNC" ) == 0 ) ? ( ( ( rowLen + grpLen - 1 ) / grpLen ) * grpLen ) : ( rowLen ) ;
		           columns = ( positions + th - 1 ) / th;
		      totalColumns = totalColumns + columns;
		       rowStartPos = rowStartPos + positions;
	}
	     totalTiles = ( totalColumns + thw - 1 ) / thw;
	 matAXT->tileN = totalTiles;
	 matAXT->lenAX = totalTiles * 2 * ths;
	if      ( (strcmp(mode, "UNC")==0) && (th==1) ) matAXT->lenSEC = totalTiles;
	else if ( (strcmp(mode, "UNC")==0) && (th!=1) ) matAXT->lenSEC = totalTiles * thw;
	else                                            matAXT->lenSEC = totalTiles * ths;
	return;
}



static void getArraysLenAXT_UNC_H1( const str_matCSR matCSR, str_matAXT * matAXT )
{
	const UIN nrows = matAXT->nrows;
	const UIN ths   = matAXT->tileHW;
	const UIN ts    = 2 * ths;
	      UIN rid, tiles, totalTiles = 0;
	 matAXT->con[0] = 0;
	for ( rid = 0; rid < nrows; rid++ )
	{
		tiles              = ( matCSR.rl[rid] + ths - 1 ) / ths;
		totalTiles         = totalTiles + tiles;
		matAXT->con[rid+1] = matAXT->con[rid] + tiles * ts;
	}
	matAXT->tileN  = totalTiles;
	matAXT->lenAX  = totalTiles * ts;
	matAXT->lenSEC = totalTiles;
	return;
}



static void getArraysLenAXT_COM_H1( const UIN ompNT, const str_matCSR matCSR, str_matAXT * matAXT )
{
	const UIN nrows = matAXT->nrows;
	const UIN ths   = matAXT->tileHW;
	      UIN rid, totalElements = 0, totalTiles;
	#pragma omp parallel for default(shared) private(rid) reduction(+:totalElements) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( rid = 0; rid < nrows; rid++ )
		totalElements = totalElements + matCSR.rl[rid];
	totalTiles     = ( totalElements + ths - 1 ) / ths;
	matAXT->tileN  =     totalTiles;
	matAXT->lenAX  = 2 * totalTiles * ths;
	matAXT->lenSEC =     totalTiles * ths;
	return;
}



static void getArraysLenAXT_COM( const UIN ompNT, const str_matCSR matCSR, str_matAXT * matAXT )
{
	const UIN nrows = matAXT->nrows;
	const UIN ths   = matAXT->tileHW * matAXT->tileH;
	      UIN rid, totalElements = 0, totalTiles;
	#pragma omp parallel for default(shared) private(rid) reduction(+:totalElements) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( rid = 0; rid < nrows; rid++ )
		totalElements = totalElements + matCSR.rl[rid];
	totalTiles     = ( totalElements + ths - 1 ) / ths;
	matAXT->tileN  =     totalTiles;
	matAXT->lenAX  = 2 * totalTiles * ths;
	matAXT->lenSEC =     totalTiles * ths;
	return;
}



static void getArraysAxSecAXT_UNC_H1( const UIN ompNT, const str_matCSR matCSR, const FPT * vec, str_matAXT * matAXT )
{
	const UIN nrows = matAXT->nrows;
	const UIN   thw = matAXT->tileHW;
	const UIN    th = matAXT->tileH;
	const UIN   ths = thw * th;
	      UIN rowID, rowLen, posAX, posSEC, posCSR, ctrEle;
	#pragma omp parallel for default(shared) private(rowID,rowLen,posAX,posSEC,posCSR,ctrEle) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( rowID = 0; rowID < nrows; rowID++ )
	{
		rowLen = matCSR.rl[rowID];
		if (rowLen>0)
		{
			posAX  = matAXT->con[rowID];
			posSEC = (posAX/(2*ths));
			ctrEle = 0;
			for ( posCSR = matCSR.row[rowID]; posCSR < matCSR.row[rowID+1]; posCSR++ )
			{
				matAXT->ax[posAX]     = matCSR.val[posCSR];
				matAXT->ax[posAX+thw] = vec[matCSR.col[posCSR]];
				matAXT->sec[posSEC]   = rowID;
				posAX++;
				ctrEle++;
				if ((ctrEle%thw)==0)
				{
					posAX = posAX + thw;
					posSEC++;
				}
			}
		}
	}
	return;
}



static void getArraysAxSecAXT_UNC( const UIN ompNT, const str_matCSR matCSR, const FPT * vec, str_matAXT * matAXT )
{
	const UIN nrows = matAXT->nrows;
	const UIN   thw = matAXT->tileHW;
	const UIN    th = matAXT->tileH;
	const UIN   ths = thw * th;
	      UIN rowID, rowLen, posAX, posSEC, posCSR, ctrEle;
	#pragma omp parallel for default(shared) private(rowID,rowLen,posAX,posSEC,posCSR,ctrEle) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( rowID = 0; rowID < nrows; rowID++ )
	{
		rowLen = matCSR.rl[rowID];
		if (rowLen>0)
		{
			posAX  = matAXT->con[rowID];
			posSEC = (posAX/(2*ths))*thw + posAX%thw;
			ctrEle = 0;
			for ( posCSR = matCSR.row[rowID]; posCSR < matCSR.row[rowID+1]; posCSR++ )
			{
				matAXT->ax[posAX]     = matCSR.val[posCSR];
				matAXT->ax[posAX+thw] = vec[matCSR.col[posCSR]];
				matAXT->sec[posSEC]   = rowID;
				posAX                 = posAX  + 2 * thw;
				ctrEle++;
				if ((ctrEle%th) == 0)
				{
					posAX = posAX + 1 - (2 * th * thw);
					posSEC++;
					if (posAX%thw==0) posAX = posAX + ((2*th)-1) * thw;
				}
			}
		}
	}
	return;
}



static void getArraysAxSecAXT_COM_H1( const UIN bs, const UIN log, const UIN ompNT, const str_matCSR matCSR, const FPT * vec, str_matAXT * matAXT )
{
	const UIN nrows = matAXT->nrows;
	const UIN   thw = matAXT->tileHW;
	      UIN rowID, rowLen, eleCtr, posCSR, bid, bco, tid, tco, posAX, posSEC, posBLK, q1, q2, offset, blk;
	#pragma omp parallel for default(shared) private(rowID,rowLen,eleCtr,posCSR,bid,bco,tid,tco,posAX,posSEC,posBLK,q1,q2,offset) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( rowID = 0; rowID < nrows; rowID++ )
	{
		rowLen = matCSR.rl[rowID];
		if (rowLen>0)
		{
			eleCtr = 0;
			for ( posCSR = matCSR.row[rowID]; posCSR < matCSR.row[rowID+1]; posCSR++ )
			{
				bid                   = ((posCSR+bs)/bs)-1;
				bco                   =   posCSR%bs;
				tid                   = ((posCSR+thw)/thw)-1;
				tco                   =  posCSR%thw;
				posAX                 = tid * 2 * thw + tco;
				posSEC                = tid     * thw + tco;
				posBLK                = bid     * bs  + bco;
				matAXT->ax[posAX]     = matCSR.val[posCSR];
				matAXT->ax[posAX+thw] = vec[matCSR.col[posCSR]];
				if ( (eleCtr==0) || (bco==0))
				{
					q1     = rowLen - eleCtr - 1;
					q2     = bs - 1 - bco;
					offset = (q1 > q2) ? q2 : q1;
					matAXT->sec[posSEC] = rowID<<log | offset;
				}
				eleCtr++;
			}
		}
	}
	return;
}



static void getArraysAxSecAXT_COM( const UIN ompNT, str_matCSR matCSR, const FPT * vec, str_matAXT * matAXT )
{
	const UIN nrows = matAXT->nrows;
	const UIN th    = matAXT->tileH;
	const UIN thw   = matAXT->tileHW;
	const UIN log   = matAXT->log;
	const UIN ths   = th * thw;
	const UIN ts    =  2 * ths;
	      UIN rid, rl, ec, pCSR, tid, fid, cid, pAX, pSEC, q1, q2, offset;
	#pragma omp parallel for default(shared) private(rid,rl,ec,pCSR,tid,fid,cid,pAX,pSEC,q1,q2,offset) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( rid = 0; rid < nrows; rid++ )
	{
		rl = matCSR.rl[rid];
		if (rl>0)
		{
			ec = 0;
			for ( pCSR = matCSR.row[rid]; pCSR < matCSR.row[rid+1]; pCSR++ )
			{
				tid  = ( (pCSR + ths) / ths ) - 1;
				fid  = pCSR % th;
				cid  = ( ( (pCSR - tid * ths) + th ) / th ) - 1;
				pAX  = tid * ts  + 2 * fid * thw + cid;
				pSEC = tid * ths +     fid * thw + cid;
				matAXT->ax[pAX]     = matCSR.val[pCSR];
				matAXT->ax[pAX+thw] = vec[matCSR.col[pCSR]];
				if ( (ec==0) || (fid==0) )
				{
					q1     = rl - ec - 1;
					q2     = th - 1 - fid;
					offset = (q1 > q2) ? q2 : q1;
					matAXT->sec[pSEC] = rid << log | offset;
				}
				ec++;
			}
		}
	}
	return;
}



static void fill_array_uin( const UIN ompNT, const UIN len, const UIN value, UIN * array )
{
	UIN i;
	#pragma omp parallel for default(shared) private(i) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for( i = 0; i < len; i++ )
	array[i] = value;
	return;
}



static str_formatData getFormatDataAXT( const UIN ompNT, const UIN bs, const UIN thw, const UIN th, const char * mode, const str_matCSR matCSR, const FPT * vec, str_matAXT * matAXT )
{
	// set AXT parameters
	matAXT->nrows  = matCSR.nrows;
	matAXT->nnz    = matCSR.nnz;
	matAXT->bs     = bs;
	matAXT->tileHW = thw;
	matAXT->tileH  = th;
	strcpy( matAXT->mode, mode );
	matAXT->lenCON = matCSR.nrows;
	   matAXT->con = (UIN *) calloc( matAXT->lenCON + 1, sizeof(UIN) ); TEST_POINTER( matAXT->con );
	UIN i;
	for ( i = 0; i < 10; i++ )
		if ( ((matAXT->tileH) >> i) == 1 ) matAXT->logTH = i;
	// get AXT arrays' length
	struct timespec t1, t2;
	double ti = 0.0, tt = 0.0, tc = 0.0;
	if (strcmp(mode,"UNC")==0)
	{
		if (th == 1)
		{
			for ( i = 0; i < NUM_ITE; i++ )
			{
				GT( t1 );
				getArraysLenAXT_UNC_H1( matCSR, matAXT );
				GT( t2 );
				ti = measure_time( t2, t1 );
				tt = tt + ti;
			}
		}
		else
		{
			for ( i = 0; i < NUM_ITE; i++ )
			{
				GT( t1 );
				getArraysLenAXT( matCSR, matAXT );
				GT( t2 );
				ti = measure_time( t2, t1 );
				tt = tt + ti;
			}
		}
	}
	else
	{
		if (th == 1)
		{
			for ( i = 0; i < NUM_ITE; i++ )
			{
				GT( t1 );
				getArraysLenAXT_COM_H1( ompNT, matCSR, matAXT );
				GT( t2 );
				ti = measure_time( t2, t1 );
				tt = tt + ti;
			}
		}
		else
		{
			for ( i = 0; i < NUM_ITE; i++ )
			{
				GT( t1 );
				getArraysLenAXT_COM( ompNT, matCSR, matAXT );
				GT( t2 );
				ti = measure_time( t2, t1 );
				tt = tt + ti;
			}
		}
	}
	ti = tt / (double) NUM_ITE;
	tc = tc + ti;
	// get arrays ax[] and sec[]
	matAXT->ax  = (FPT *) _mm_malloc( matAXT->lenAX  * sizeof(FPT), 64 );  TEST_POINTER( matAXT->ax  ); fill_array    ( ompNT, matAXT->lenAX,  0.0, matAXT->ax  );
	matAXT->sec = (UIN *) _mm_malloc( matAXT->lenSEC * sizeof(UIN), 64 );  TEST_POINTER( matAXT->sec ); fill_array_uin( ompNT, matAXT->lenSEC,   0, matAXT->sec );
	tt = 0.0;
	char buffer[48];
	if (strcmp(mode,"UNC")==0)
	{
		if (th==1)
		{
			for ( i = 0; i < NUM_ITE; i++ )
			{
				GT( t1 );
				getArraysAxSecAXT_UNC_H1( ompNT, matCSR, vec, matAXT );
				GT( t2 );
				ti = measure_time( t2, t1 );
				tt = tt + ti;
			}
			char THW[5]; sprintf( THW, "%d", thw );
			strcpy( buffer, "f_axt_unc_h1_hw" );
			strcat( buffer, THW );
		}
		else
		{
			for ( i = 0; i < NUM_ITE; i++ )
			{
				GT( t1 );
				getArraysAxSecAXT_UNC( ompNT, matCSR, vec, matAXT );
				GT( t2 );
				ti = measure_time( t2, t1 );
				tt = tt + ti;
			}
			char TH[5];  sprintf( TH,  "%d", th  );
			char THW[5]; sprintf( THW, "%d", thw );
			strcpy( buffer, "f_axt_unc_h" );
			strcat( buffer, TH );
			strcat( buffer, "_hw" );
			strcat( buffer, THW );
		}
	}
	else
	{
		if (th==1)
		{
			for ( i = 1; i < 10; i++ )
			{
				if ((bs>>i) == 1)
				{
					matAXT->log = i;
					break;
				}
			}
			for ( i = 0; i < NUM_ITE; i++ )
			{
				GT( t1 );
				getArraysAxSecAXT_COM_H1( bs, matAXT->log, ompNT, matCSR, vec, matAXT );
				GT( t2 );
				ti = measure_time( t2, t1 );
				tt = tt + ti;
			}
			char THW[5]; sprintf( THW, "%d", thw );
			char BS[5];  sprintf( BS, "%d", bs );
			strcpy( buffer, "f_axt_com_h1_hw" );
			strcat( buffer, THW );
			strcat( buffer, "_bs" );
			strcat( buffer, BS );
		}
		else
		{
			for ( i = 1; i < 10; i++ )
			{
				if ((th>>i) == 1)
				{
					matAXT->log = i;
					break;
				}
			}
			for ( i = 0; i < NUM_ITE; i++ )
			{
				GT( t1 );
				getArraysAxSecAXT_COM( ompNT, matCSR, vec, matAXT );
				GT( t2 );
				ti = measure_time( t2, t1 );
				tt = tt + ti;
			}
			char TH[5];  sprintf( TH,  "%d", th  );
			char THW[5]; sprintf( THW, "%d", thw );
			strcpy( buffer, "f_axt_com_h" );
			strcat( buffer, TH );
			strcat( buffer, "_hw" );
			strcat( buffer, THW );
		}
	}
	ti = tt / (double) NUM_ITE;
	tc = tc + ti;
	// AXTC specific name
	str_formatData fd;
	strcpy( fd.name, buffer );
	// AXTC memory footprint
	fd.mfp =          (double) ( matAXT->lenAX  * sizeof(FPT) ); // ax
	fd.mfp = fd.mfp + (double) ( matAXT->lenSEC * sizeof(UIN) ); // sec
	// AXTC occupancy ( beta )
	fd.beta = ( (double) matAXT->nnz / (double) (matAXT->lenAX >> 1) );
	// AXTC conversion time
	fd.ct = tc;
	return( fd );
}



static void int_axt_unc_h1( const UIN ompNT, const UIN tn, const UIN thw, const FPT * ax, const UIN * rwp, FPT * y )
{
	const UIN stride = 2 * thw;
	      UIN tid, posAX, rowID;
	      FPT red;
	  __m512d vtMat, vtVec, vtPro;
	#pragma omp parallel for default(shared) private(tid,posAX,vtMat,vtVec,vtPro,red,rowID) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( tid = 0; tid < tn; tid++ )
	{
		posAX    = tid * stride;
		vtMat    = _mm512_load_pd( &ax[posAX]       );
		vtVec    = _mm512_load_pd( &ax[posAX + thw] );
		vtPro    = _mm512_mul_pd( vtMat, vtVec );
		red      = _mm512_reduce_add_pd( vtPro );
		rowID    = rwp[tid];
		#pragma omp atomic
		y[rowID] = y[rowID] + red;
	}
	return;
}



static str_res test_int_axt_unc_h1( const UIN ompNT, const str_matAXT matAXT, const FPT * ref )
{
	//
	const UIN tileN  = matAXT.tileN;
	const UIN tileHW = matAXT.tileHW;
	const UIN nrows  = matAXT.nrows;
	const UIN nnz    = matAXT.nnz;
	// timed iterations
	double ti = 0.0, tt = 0.0;
	struct timespec t1, t2;
	FPT * res = (FPT *) calloc( nrows, sizeof(FPT) ); TEST_POINTER( res );
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		fill_array( ompNT, nrows, 0, res );
		GT( t1 );
		int_axt_unc_h1( ompNT, tileN, tileHW, matAXT.ax, matAXT.sec, res );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	char THW[5]; sprintf( THW, "%d", tileHW );
	char buffer[48];
	strcpy( buffer, "p_axt_unc_h1_hw" );
	strcat( buffer, THW );
	// store results
	str_res sr;
	strcpy( sr.name, buffer );
	sr.et    = tt / (double) NUM_ITE;
	sr.ot    = 0.0;
	sr.flops = ( 2.0 * ( (double) nnz ) ) / sr.et;
	get_errors( nrows, ref, res, &(sr.sErr) );
	free( res );
	return( sr );
}


static void int_axt_unc( const UIN ompNT, const UIN tn, const UIN th, const UIN thw, const FPT * ax, const UIN * rwp, FPT * y )
{
	const UIN stride = 2 * thw;
	const UIN ts     = th * stride;
	      UIN tid, posAX, posRWP, rowID, i;
	      FPT tmp[thw];
	      FPT red;
	  __m512d vtMat, vtVec, vtPro, vtSum;
	  __m512i vtRid;
	#pragma omp parallel for default(shared) private(tid,vtSum,posAX,vtMat,vtVec,vtPro,tmp,i,posRWP,rowID) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( tid = 0; tid < tn; tid++ )
	{
		vtSum = _mm512_setzero_pd();
		for ( posAX = tid * ts; posAX < (tid + 1) * ts; posAX = posAX + stride )
		{
			vtMat = _mm512_load_pd( &ax[posAX]       );
			vtVec = _mm512_load_pd( &ax[posAX + thw] );
			vtPro = _mm512_mul_pd( vtMat, vtVec );
			vtSum = _mm512_add_pd( vtSum, vtPro );
		}
		_mm512_store_pd( tmp, vtSum );
		for ( i = 0, posRWP = tid * thw; i < thw; i++, posRWP++ )
		{
			rowID    = rwp[posRWP];
			y[rowID] = y[rowID] + tmp[i];
		}
	}
	return;
}



static str_res test_int_axt_unc( const UIN ompNT, const str_matAXT matAXT, const FPT * ref )
{
	//
	const UIN tileNum = matAXT.tileN;
	const UIN tileH   = matAXT.tileH;
	const UIN tileHW  = matAXT.tileHW;
	const UIN nrows   = matAXT.nrows;
	const UIN nnz     = matAXT.nnz;
	// timed iterations
	double ti = 0.0, tt = 0.0;
	struct timespec t1, t2;
	FPT * res = (FPT *) _mm_malloc( nrows * sizeof(FPT), 64 ); TEST_POINTER( res );
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		fill_array( ompNT, nrows, 0, res );
		GT( t1 );
		int_axt_unc( ompNT, tileNum, tileH, tileHW, matAXT.ax, matAXT.sec, res );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	char TH[5];      sprintf( TH,   "%d", tileH  );
	char THW[5];     sprintf( THW,  "%d", tileHW );
	char buffer[48]; strcpy( buffer, "p_axt_unc_h" );
	strcat( buffer, TH );
	strcat( buffer, "_hw" );
	strcat( buffer, THW );
	// store results
	str_res sr;
	strcpy( sr.name, buffer );
	sr.et    = tt / (double) NUM_ITE;
	sr.ot    = 0.0;
	sr.flops = ( 2.0 * ( (double) nnz ) ) / sr.et;
	get_errors( nrows, ref, res, &(sr.sErr) );
	_mm_free( res );
	return( sr );
}



static __m512d inc_scan8( __m512d vtVal )
{
	__m512i  vtIdx = _mm512_set_epi64( 6, 6, 4, 4, 2, 2, 0, 0 );
	__m512d  vtAux = _mm512_permutexvar_pd( vtIdx, vtVal );
	__mmask8 cmask = 0xAA;
	__m512d  vtRed = _mm512_mask_add_pd( vtVal, cmask, vtVal, vtAux );
	         vtIdx = _mm512_set_epi64( 5, 6, 5, 4, 1, 2, 1, 0 );
	         vtAux = _mm512_permutexvar_pd( vtIdx, vtRed );
	         cmask = 0x88;
	         vtRed = _mm512_mask_add_pd( vtRed, cmask, vtRed, vtAux );
	         vtIdx = _mm512_set_epi64( 3, 6, 5, 4, 3, 2, 1, 0 );
	         vtAux = _mm512_permutexvar_pd( vtIdx, vtRed );
	         cmask = 0x80;
	         vtRed = _mm512_mask_add_pd( vtRed, cmask, vtRed, vtAux );
	         vtIdx = _mm512_set_epi64( 7, 6, 3, 4, 3, 2, 1, 0 );
	         vtAux = _mm512_permutexvar_pd( vtIdx, vtRed );
	         cmask = 0x20;
	         vtRed = _mm512_mask_add_pd( vtRed, cmask, vtRed, vtAux );
	         vtIdx = _mm512_set_epi64( 7, 5, 5, 3, 3, 1, 1, 0 );
	         vtAux = _mm512_permutexvar_pd( vtIdx, vtRed );
	         cmask = 0x54;
	         vtRed = _mm512_mask_add_pd( vtRed, cmask, vtRed, vtAux );
	return( vtRed );
}



static void int_axt_com_h1_bs64( const UIN ompNT, const UIN bn, const UIN lenRWP, const FPT * ax, const UIN * rwp, FPT * y )
{
	      UIN bid, blkOff, tid, posAX, posBLK, ep, ro, r, o;
	  __m512d vtMat, vtVec, vtVal, vtRed, vtAcu;
	      FPT v;
	      FPT blk1[64];
	      FPT blk2[64];
	      FPT blk3[8];
	#pragma omp parallel for default(shared) private(bid,blkOff,tid,posAX,posBLK,ep,ro,r,o,vtMat,vtVec,vtVal,vtRed,vtAcu,v,blk1,blk2,blk3) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( bid = 0; bid < bn; bid++ )
	{
		blkOff = bid * 128;
		for ( tid = 0; tid < 8; tid++ )
		{
			posAX     = blkOff + tid * 16;
			vtMat     = _mm512_load_pd( &ax[posAX]     );
			vtVec     = _mm512_load_pd( &ax[posAX + 8] );
			vtVal     = _mm512_mul_pd( vtMat, vtVec );
			posBLK    = tid *  8;
			_mm512_store_pd( &blk1[posBLK], vtVal );
			vtRed     = inc_scan8( vtVal );
			_mm512_store_pd( &blk2[posBLK], vtRed );
			blk3[tid] = blk2[posBLK+7];
		}
		vtVal = _mm512_load_pd( blk3 );
		vtRed = inc_scan8( vtVal );
		_mm512_store_pd( blk3, vtRed );
		for ( tid = 1; tid < 8; tid++ )
		{
			posBLK = tid * 8;
			vtAcu  = _mm512_set1_pd( blk3[tid-1]   );
			vtVal  = _mm512_load_pd( &blk2[posBLK] );
			vtRed  = _mm512_add_pd( vtAcu, vtVal );
			_mm512_store_pd( &blk2[posBLK], vtRed );
		}
		blkOff = bid * 64;
		ep     = blkOff + 64;
		ep     = (ep>lenRWP) ? lenRWP : ep ;
		#pragma omp parallel for default(shared) private(posBLK,tid,ro,r,o,v) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
		for ( posBLK = blkOff, tid = 0; posBLK < ep; posBLK++, tid++ )
		{
			ro = rwp[posBLK];
			if (ro!=0)
			{
				r = ro >> 6;
				o = ro & 63;
				v = blk2[tid+o] - blk2[tid] + blk1[tid];
				#pragma omp atomic
				y[r] = y[r] + v;
			}
		}
	}
	return;
}



static void int_axt_com_h1_bs512( const UIN ompNT, const UIN bn, const UIN lenRWP, const FPT * ax, const UIN * rwp, FPT * y )
{
	      UIN bid, blkOff, tid, posAX, posBLK, ep, ro, r, o;
	  __m512d vtMat, vtVec, vtVal, vtRed, vtAcu;
	      FPT v;
	      FPT blk1[512];
	      FPT blk2[512];
	      FPT blk3[64];
	      FPT blk4[8];
	#pragma omp parallel for default(shared) private(bid,blkOff,tid,posAX,posBLK,ep,ro,r,o,vtMat,vtVec,vtVal,vtRed,vtAcu,v,blk1,blk2,blk3,blk4) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( bid = 0; bid < bn; bid++ )
	{
		blkOff = bid * 1024;
		for ( tid = 0; tid < 64; tid++ )
		{
			posAX     = blkOff + tid * 16;
			vtMat     = _mm512_load_pd( &ax[posAX]     );
			vtVec     = _mm512_load_pd( &ax[posAX + 8] );
			vtVal     = _mm512_mul_pd( vtMat, vtVec );
			posBLK    = tid *  8;
			_mm512_store_pd( &blk1[posBLK], vtVal );
			vtRed     = inc_scan8( vtVal );
			_mm512_store_pd( &blk2[posBLK], vtRed );
			blk3[tid] = blk2[posBLK+7];
		}
		for ( tid = 0; tid < 8; tid++ )
		{
			posBLK  = tid *  8;
			vtVal   = _mm512_load_pd( &blk3[posBLK] );
			vtRed   = inc_scan8( vtVal );
			_mm512_store_pd( &blk3[posBLK], vtRed );
			blk4[tid] = blk3[posBLK+7];
		}
		vtVal = _mm512_load_pd( blk4 );
		vtRed = inc_scan8( vtVal );
		_mm512_store_pd( blk4, vtRed );
		for ( tid = 1; tid < 8; tid++ )
		{
			posBLK = tid * 8;
			vtAcu  = _mm512_set1_pd( blk4[tid-1]   );
			vtVal  = _mm512_load_pd( &blk3[posBLK] );
			vtRed  = _mm512_add_pd( vtAcu, vtVal );
			_mm512_store_pd( &blk3[posBLK], vtRed );
		}
		for ( tid = 1; tid < 64; tid++ )
		{
			posBLK = tid * 8;
			vtAcu  = _mm512_set1_pd( blk3[tid-1]   );
			vtVal  = _mm512_load_pd( &blk2[posBLK] );
			vtRed  = _mm512_add_pd( vtAcu, vtVal );
			_mm512_store_pd( &blk2[posBLK], vtRed );
		}
		blkOff = bid * 512;
		ep     = blkOff + 512;
		ep     = (ep>lenRWP) ? lenRWP : ep ;
		#pragma omp parallel for default(shared) private(posBLK,tid,ro,r,o,v) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
		for ( posBLK = blkOff, tid = 0; posBLK < ep; posBLK++, tid++ )
		{
			ro = rwp[posBLK];
			if (ro!=0)
			{
				r = ro >> 9;
				o = ro & 511;
				v = blk2[tid+o] - blk2[tid] + blk1[tid];
				#pragma omp atomic
				y[r] = y[r] + v;
			}
		}
	}
	return;
}





static str_res test_int_axt_com_h1_bs64( const UIN ompNT, const str_matAXT matAXT, const FPT * ref )
{
	//
	const UIN nrows  = matAXT.nrows;
	const UIN nnz    = matAXT.nnz;
	const UIN lenAX  = matAXT.lenAX;
	const UIN lenSEC = matAXT.lenSEC;
	const UIN log    = matAXT.log;
	const UIN tn     = matAXT.tileN;
	const UIN thw    = matAXT.tileHW;
	const UIN bs     = matAXT.bs;
	const UIN ts     = 2 * thw;
	const UIN bn     = ( (tn * thw) + bs - 1 ) / bs;
	const UIN tpb    = bs / thw;
	FPT * res = (FPT *) _mm_malloc( nrows   * sizeof(FPT), 64 ); TEST_POINTER( res );
	// timed iterations
	double ti = 0.0, tt = 0.0;
	struct timespec t1, t2;
	UIN i;
	const UIN newLenAX  = 2 * bn * bs; FPT * ax  = (FPT *) _mm_malloc( newLenAX  * sizeof(FPT), 64 ); TEST_POINTER( ax  );
	const UIN newLenSEC =     bn * bs; UIN * sec = (UIN *)     calloc( newLenSEC, sizeof(UIN) );      TEST_POINTER( sec );
	fill_array( ompNT, newLenAX, 0, ax );
	cblas_dcopy( lenAX, matAXT.ax, 1, ax, 1 );
	memcpy( sec, matAXT.sec, lenSEC * sizeof(UIN) );
	for ( i = 0; i < NUM_ITE; i++ )
	{
		fill_array( ompNT, nrows, 0, res );
		GT( t1 );
		int_axt_com_h1_bs64( ompNT, bn, lenSEC, ax, sec, res );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	_mm_free( ax );
	free( sec );
	// store results
	char THW[5]; sprintf( THW, "%d", thw );
	char BS[5];  sprintf( BS,  "%d", bs  );
	char buffer[48];
	strcpy( buffer, "p_axt_com_h1_hw" );
	strcat( buffer, THW );
	strcat( buffer, "_bs" );
	strcat( buffer, BS );
	// store results
	str_res sr;
	strcpy( sr.name, buffer );
	sr.et    = tt / (double) NUM_ITE;
	sr.ot    = 0.0;
	sr.flops = ( 2.0 * ( (double) nnz ) ) / sr.et;
	get_errors( nrows, ref, res, &(sr.sErr) );
	_mm_free( res );
	return( sr );
}



static str_res test_int_axt_com_h1_bs512( const UIN ompNT, const str_matAXT matAXT, const FPT * ref )
{
	//
	const UIN nrows  = matAXT.nrows;
	const UIN nnz    = matAXT.nnz;
	const UIN lenAX  = matAXT.lenAX;
	const UIN lenSEC = matAXT.lenSEC;
	const UIN log    = matAXT.log;
	const UIN tn     = matAXT.tileN;
	const UIN thw    = matAXT.tileHW;
	const UIN bs     = matAXT.bs;
	const UIN ts     = 2 * thw;
	const UIN bn     = ( (tn * thw) + bs - 1 ) / bs;
	const UIN tpb    = bs / thw;
	FPT * res = (FPT *) _mm_malloc( nrows   * sizeof(FPT), 64 ); TEST_POINTER( res );
	// timed iterations
	double ti = 0.0, tt = 0.0;
	struct timespec t1, t2;
	UIN i;
	const UIN newLenAX  = 2 * bn * bs; FPT * ax  = (FPT *) _mm_malloc( newLenAX  * sizeof(FPT), 64 ); TEST_POINTER( ax  );
	const UIN newLenSEC =     bn * bs; UIN * sec = (UIN *)     calloc( newLenSEC, sizeof(UIN) );      TEST_POINTER( sec );
	fill_array( ompNT, newLenAX, 0, ax );
	cblas_dcopy( lenAX, matAXT.ax, 1, ax, 1 );
	memcpy( sec, matAXT.sec, lenSEC * sizeof(UIN) );
	for ( i = 0; i < NUM_ITE; i++ )
	{
		fill_array( ompNT, nrows, 0, res );
		GT( t1 );
		int_axt_com_h1_bs512( ompNT, bn, lenSEC, ax, sec, res );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	_mm_free( ax );
	free( sec );
	// store results
	char THW[5]; sprintf( THW, "%d", thw );
	char BS[5];  sprintf( BS,  "%d", bs  );
	char buffer[48];
	strcpy( buffer, "p_axt_com_h1_hw" );
	strcat( buffer, THW );
	strcat( buffer, "_bs" );
	strcat( buffer, BS );
	// store results
	str_res sr;
	strcpy( sr.name, buffer );
	sr.et    = tt / (double) NUM_ITE;
	sr.ot    = 0.0;
	sr.flops = ( 2.0 * ( (double) nnz ) ) / sr.et;
	get_errors( nrows, ref, res, &(sr.sErr) );
	_mm_free( res );
	return( sr );
}



static void int_axt_com( const UIN ompNT, const UIN tn, const UIN th, const UIN log, const UIN thw, const FPT * ax, const UIN * rwp, FPT * y )
{
	const UIN ths = th * thw;
	const UIN ts  =  2 * ths;
	      UIN tid, off, f, pAX, pBLK, ro, r, o;
	  __m512d vtMat, vtVec, vtVal, vtAcu;
	      FPT blk1[th*thw];
	      FPT blk2[th*thw];
	      FPT v;
	#pragma omp parallel for default(shared) private(tid,off,f,pAX,pBLK,ro,r,o,vtMat,vtVec,vtVal,vtAcu,v,blk1,blk2) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
	for ( tid = 0; tid < tn; tid++ )
	{
		off = tid * ts;
		vtAcu = _mm512_setzero_pd();
		for ( f = 0; f < th; f++ )
		{
			pAX   = off + f * 16;
			vtMat = _mm512_load_pd( &ax[pAX]     );
			vtVec = _mm512_load_pd( &ax[pAX + 8] );
			vtVal = _mm512_mul_pd( vtMat, vtVec );
			pBLK  = f * 8;
			_mm512_store_pd( &blk1[pBLK], vtVal );
			vtAcu = _mm512_add_pd( vtAcu, vtVal );
			_mm512_store_pd( &blk2[pBLK], vtAcu );
		}
		off = tid * ths;
		#pragma omp parallel for default(shared) private(f,ro,r,o,v) num_threads(ompNT) schedule(OMP_SCH) if(_OPENMP)
		for ( f = 0; f < ths; f++ )
		{
			ro = rwp[off + f];
			if (ro != 0)
			{
				r    = ro >> log;
				o    = ro & (th-1);
				v    = blk2[f+(o*thw)] - blk2[f] + blk1[f];
				#pragma omp atomic
				y[r] = y[r] + v;
			}
		}
	}
	return;
}



static str_res test_int_axt_com( const UIN ompNT, const str_matAXT matAXT, const FPT * ref )
{
	//
	const UIN nrows  = matAXT.nrows;
	const UIN nnz    = matAXT.nnz;
	const UIN tn     = matAXT.tileN;
	const UIN th     = matAXT.tileH;
	const UIN thw    = matAXT.tileHW;
	const UIN log    = matAXT.log;
	FPT * res = (FPT *) _mm_malloc( nrows   * sizeof(FPT), 64 ); TEST_POINTER( res );
	// timed iterations
	double ti = 0.0, tt = 0.0;
	struct timespec t1, t2;
	UIN i;
	for ( i = 0; i < NUM_ITE; i++ )
	{
		fill_array( ompNT, nrows, 0, res );
		GT( t1 );
		int_axt_com( ompNT, tn, th, log, thw, matAXT.ax, matAXT.sec, res );
		GT( t2 );
		ti = measure_time( t2, t1 );
		tt = tt + ti;
	}
	// store results
	char TH[5];  sprintf( TH,  "%d", th );
	char THW[5]; sprintf( THW, "%d", thw );
	char buffer[48];
	strcpy( buffer, "p_axt_com_h" );
	strcat( buffer, TH );
	strcat( buffer, "_hw" );
	strcat( buffer, THW );
	// store results
	str_res sr;
	strcpy( sr.name, buffer );
	sr.et    = tt / (double) NUM_ITE;
	sr.ot    = 0.0;
	sr.flops = ( 2.0 * ( (double) nnz ) ) / sr.et;
	get_errors( nrows, ref, res, &(sr.sErr) );
	_mm_free( res );
	return( sr );
}



*/

#endif



int main( int argc, char ** argv )
{
	// check input arguments
	str_inputArgs sia = checkArgs( argc, argv );

	// print run settings
	printRunSettings( sia );

	// count available GPU devices
	int cudaDeviceCounter;
	getCudaDeviceCounter( &cudaDeviceCounter );

	// print GPUs' info
	if ( cudaDeviceCounter > 0 )
	{
		int cudaDeviceID;
		for ( cudaDeviceID = 0; cudaDeviceID < cudaDeviceCounter; cudaDeviceID++ )
			printCudaDeviceProperties( cudaDeviceID );
		cudaDeviceID = DEVICE;
		HANDLE_CUDA_ERROR( hipSetDevice( cudaDeviceID) );
		HANDLE_CUDA_ERROR( hipGetDevice(&cudaDeviceID) );
		printf( "cudaDeviceSelected:                               %d <-------------------\n", cudaDeviceID );
	}

	// CSR format  ------------------------------------------------------------------------------------------------------------------
	// read matrix in CSR format
	str_matCSR matCSR = matrixReading( sia.matFileName );
	// print matrix's statistics
	printMatrixStats( sia.matFileName, &matCSR );

	// get memory footprint, occupancy (beta) and conversion time
	str_formatData fd01 = getFormatDataCSR( &matCSR );

	// CSR format  ------------------------------------------------------------------------------------------------------------------
	// init vectors to perform SpMV multiplication and check errors (spM * vr = yr)
	FPT * vr = (FPT *) calloc( matCSR.nrows, sizeof(FPT) ); TEST_POINTER( vr );
	init_vec( sia.ompMT, matCSR.nrows, vr );
	FPT * yr = (FPT *) calloc( matCSR.nrows,  sizeof(FPT) ); TEST_POINTER( yr );
	ncsr( sia.ompMT, matCSR, vr, yr );
	// test CSR kernels
	str_res sr01 = test_ncsr( sia.ompMT, matCSR, vr, yr );
	str_res sr02 = test_gcsr( sia.cbs, matCSR, vr, yr );
	str_res sr03 = test_gcucsr( matCSR, vr, yr );
	// CSR format  ------------------------------------------------------------------------------------------------------------------

	// K1 format  -------------------------------------------------------------------------------------------------------------------
	str_matK1 matK1; str_formatData fd02 = getFormatDataK1( CHUNK_SIZE, matCSR, vr, &matK1 );
	str_res sr04 = test_gk1( sia.cbs, matK1, vr, yr );
	// K1 format  -------------------------------------------------------------------------------------------------------------------

	// AXC format  ------------------------------------------------------------------------------------------------------------------
	str_matAXC matAXC; str_formatData fd03 = getFormatDataAXC( sia.ompMT, 32, matCSR, vr, &matAXC );
	str_res sr05 = test_gaxc( sia.cbs, matAXC, yr );
	// AXC format  ------------------------------------------------------------------------------------------------------------------

/*
	// AXT format  ------------------------------------------------------------------------------------------------------------------
	str_matAXT matAXT1;  str_formatData fd04 = getFormatDataAXT( sia.ompMT,   64, 8,   1, "UNC", matCSR, vr, &matAXT1 );
	str_matAXT matAXT2;  str_formatData fd05 = getFormatDataAXT( sia.ompMT,   64, 8,   4, "UNC", matCSR, vr, &matAXT2 );
	str_matAXT matAXT3;  str_formatData fd06 = getFormatDataAXT( sia.ompMT,   64, 8,   8, "UNC", matCSR, vr, &matAXT3 );
	str_matAXT matAXT4;  str_formatData fd07 = getFormatDataAXT( sia.ompMT,   64, 8,  12, "UNC", matCSR, vr, &matAXT4 );
	str_matAXT matAXT5;  str_formatData fd08 = getFormatDataAXT( sia.ompMT,   64, 8,  16, "UNC", matCSR, vr, &matAXT5 );
	str_matAXT matAXT6;  str_formatData fd09 = getFormatDataAXT( sia.ompMT,   64, 8,  20, "UNC", matCSR, vr, &matAXT6 );
	str_matAXT matAXT7;  str_formatData fd10 = getFormatDataAXT( sia.ompMT,   64, 8,  24, "UNC", matCSR, vr, &matAXT7 );
	str_matAXT matAXT8;  str_formatData fd11 = getFormatDataAXT( sia.ompMT,   64, 8,  28, "UNC", matCSR, vr, &matAXT8 );
	str_matAXT matAXT9;  str_formatData fd12 = getFormatDataAXT( sia.ompMT,   64, 8,  32, "UNC", matCSR, vr, &matAXT9 );
	str_matAXT matAXT10; str_formatData fd13 = getFormatDataAXT( sia.ompMT,   64, 8,  36, "UNC", matCSR, vr, &matAXT10 );
	str_matAXT matAXT11; str_formatData fd14 = getFormatDataAXT( sia.ompMT,   64, 8,  40, "UNC", matCSR, vr, &matAXT11 );
	str_matAXT matAXT12; str_formatData fd15 = getFormatDataAXT( sia.ompMT,   64, 8,   1, "COM", matCSR, vr, &matAXT12 );
	str_matAXT matAXT13; str_formatData fd16 = getFormatDataAXT( sia.ompMT,  512, 8,   1, "COM", matCSR, vr, &matAXT13 );
	str_matAXT matAXT14; str_formatData fd17 = getFormatDataAXT( sia.ompMT,   64, 8,   8, "COM", matCSR, vr, &matAXT14 );
	str_matAXT matAXT15; str_formatData fd18 = getFormatDataAXT( sia.ompMT,   64, 8,  16, "COM", matCSR, vr, &matAXT15 );
	str_matAXT matAXT16; str_formatData fd19 = getFormatDataAXT( sia.ompMT,   64, 8,  32, "COM", matCSR, vr, &matAXT16 );
	str_matAXT matAXT17; str_formatData fd20 = getFormatDataAXT( sia.ompMT,   64, 8,  64, "COM", matCSR, vr, &matAXT17 );
	str_matAXT matAXT18; str_formatData fd21 = getFormatDataAXT( sia.ompMT,   64, 8, 128, "COM", matCSR, vr, &matAXT18 );
	str_matAXT matAXT19; str_formatData fd22 = getFormatDataAXT( sia.ompMT,   64, 8, 256, "COM", matCSR, vr, &matAXT19 );
	str_matAXT matAXT20; str_formatData fd23 = getFormatDataAXT( sia.ompMT,   64, 8, 512, "COM", matCSR, vr, &matAXT20 );
	str_res sr06 = test_int_axt_unc_h1( sia.ompMT, matAXT1, yr );
	str_res sr07 = test_int_axt_unc( sia.ompMT, matAXT2, yr );
	str_res sr08 = test_int_axt_unc( sia.ompMT, matAXT3, yr );
	str_res sr09 = test_int_axt_unc( sia.ompMT, matAXT4, yr );
	str_res sr10 = test_int_axt_unc( sia.ompMT, matAXT5, yr );
	str_res sr11 = test_int_axt_unc( sia.ompMT, matAXT6, yr );
	str_res sr12 = test_int_axt_unc( sia.ompMT, matAXT7, yr );
	str_res sr13 = test_int_axt_unc( sia.ompMT, matAXT8, yr );
	str_res sr14 = test_int_axt_unc( sia.ompMT, matAXT9, yr );
	str_res sr15 = test_int_axt_unc( sia.ompMT, matAXT10, yr );
	str_res sr16 = test_int_axt_unc( sia.ompMT, matAXT11, yr );
	str_res sr17 = test_int_axt_com_h1_bs64( sia.ompMT, matAXT12, yr );
	str_res sr18 = test_int_axt_com_h1_bs512( sia.ompMT, matAXT13, yr );
	str_res sr19 = test_int_axt_com( sia.ompMT, matAXT14, yr );
	str_res sr20 = test_int_axt_com( sia.ompMT, matAXT15, yr );
	str_res sr21 = test_int_axt_com( sia.ompMT, matAXT16, yr );
	str_res sr22 = test_int_axt_com( sia.ompMT, matAXT17, yr );
	str_res sr23 = test_int_axt_com( sia.ompMT, matAXT18, yr );
	str_res sr24 = test_int_axt_com( sia.ompMT, matAXT19, yr );
	str_res sr25 = test_int_axt_com( sia.ompMT, matAXT20, yr );
	// AXT format  ------------------------------------------------------------------------------------------------------------------
*/
	HDL; printf( "formats' data\n" ); HDL;
	printf( "%25s %20s %10s %20s\n", "format", "memory [Mbytes]", "occupancy", "convTime [s]" );
	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd01.name, ( fd01.mfp * 1e-6 ), fd01.beta, fd01.ct ); fflush(stdout);
	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd02.name, ( fd02.mfp * 1e-6 ), fd02.beta, fd02.ct ); fflush(stdout);
	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd03.name, ( fd03.mfp * 1e-6 ), fd03.beta, fd03.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd04.name, ( fd04.mfp * 1e-6 ), fd04.beta, fd04.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd05.name, ( fd05.mfp * 1e-6 ), fd05.beta, fd05.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd06.name, ( fd06.mfp * 1e-6 ), fd06.beta, fd06.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd07.name, ( fd07.mfp * 1e-6 ), fd07.beta, fd07.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd08.name, ( fd08.mfp * 1e-6 ), fd08.beta, fd08.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd09.name, ( fd09.mfp * 1e-6 ), fd09.beta, fd09.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd10.name, ( fd10.mfp * 1e-6 ), fd10.beta, fd10.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd11.name, ( fd11.mfp * 1e-6 ), fd11.beta, fd11.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd12.name, ( fd12.mfp * 1e-6 ), fd12.beta, fd12.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd13.name, ( fd13.mfp * 1e-6 ), fd13.beta, fd13.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd14.name, ( fd14.mfp * 1e-6 ), fd14.beta, fd14.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd15.name, ( fd15.mfp * 1e-6 ), fd15.beta, fd15.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd16.name, ( fd16.mfp * 1e-6 ), fd16.beta, fd16.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd17.name, ( fd17.mfp * 1e-6 ), fd17.beta, fd17.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd18.name, ( fd18.mfp * 1e-6 ), fd18.beta, fd18.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd19.name, ( fd19.mfp * 1e-6 ), fd19.beta, fd19.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd20.name, ( fd20.mfp * 1e-6 ), fd20.beta, fd20.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd21.name, ( fd21.mfp * 1e-6 ), fd21.beta, fd21.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd22.name, ( fd22.mfp * 1e-6 ), fd22.beta, fd22.ct ); fflush(stdout);
//	printf( "%25s %20.2lf %10.2lf %20.6lf\n", fd23.name, ( fd23.mfp * 1e-6 ), fd23.beta, fd23.ct ); fflush(stdout);

	HDL; printf( "SpMV kernels' results\n" ); HDL;
	printf( "%25s %15s %8s %15s %13s %13s %10s\n", "kernel", "exeTime [s]", "Gflops", "ordTime [s]", "errAbs", "errRel", "rowInd" );
	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr01.name, sr01.et, ( sr01.flops * 1e-9 ), sr01.ot, sr01.sErr.aErr, sr01.sErr.rErr, sr01.sErr.pos ); fflush(stdout);
	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr02.name, sr02.et, ( sr02.flops * 1e-9 ), sr02.ot, sr02.sErr.aErr, sr02.sErr.rErr, sr02.sErr.pos ); fflush(stdout);
	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr03.name, sr03.et, ( sr03.flops * 1e-9 ), sr03.ot, sr03.sErr.aErr, sr03.sErr.rErr, sr03.sErr.pos ); fflush(stdout);
	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr04.name, sr04.et, ( sr04.flops * 1e-9 ), sr04.ot, sr04.sErr.aErr, sr04.sErr.rErr, sr04.sErr.pos ); fflush(stdout);
	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr05.name, sr05.et, ( sr05.flops * 1e-9 ), sr05.ot, sr05.sErr.aErr, sr05.sErr.rErr, sr05.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr06.name, sr06.et, ( sr06.flops * 1e-9 ), sr06.ot, sr06.sErr.aErr, sr06.sErr.rErr, sr06.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr07.name, sr07.et, ( sr07.flops * 1e-9 ), sr07.ot, sr07.sErr.aErr, sr07.sErr.rErr, sr07.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr08.name, sr08.et, ( sr08.flops * 1e-9 ), sr08.ot, sr08.sErr.aErr, sr08.sErr.rErr, sr08.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr09.name, sr09.et, ( sr09.flops * 1e-9 ), sr09.ot, sr09.sErr.aErr, sr09.sErr.rErr, sr09.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr10.name, sr10.et, ( sr10.flops * 1e-9 ), sr10.ot, sr10.sErr.aErr, sr10.sErr.rErr, sr10.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr11.name, sr11.et, ( sr11.flops * 1e-9 ), sr11.ot, sr11.sErr.aErr, sr11.sErr.rErr, sr11.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr12.name, sr12.et, ( sr12.flops * 1e-9 ), sr12.ot, sr12.sErr.aErr, sr12.sErr.rErr, sr12.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr13.name, sr13.et, ( sr13.flops * 1e-9 ), sr13.ot, sr13.sErr.aErr, sr13.sErr.rErr, sr13.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr14.name, sr14.et, ( sr14.flops * 1e-9 ), sr14.ot, sr14.sErr.aErr, sr14.sErr.rErr, sr14.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr15.name, sr15.et, ( sr15.flops * 1e-9 ), sr15.ot, sr15.sErr.aErr, sr15.sErr.rErr, sr15.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr16.name, sr16.et, ( sr16.flops * 1e-9 ), sr16.ot, sr16.sErr.aErr, sr16.sErr.rErr, sr16.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr17.name, sr17.et, ( sr17.flops * 1e-9 ), sr17.ot, sr17.sErr.aErr, sr17.sErr.rErr, sr17.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr18.name, sr18.et, ( sr18.flops * 1e-9 ), sr18.ot, sr18.sErr.aErr, sr18.sErr.rErr, sr18.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr19.name, sr19.et, ( sr19.flops * 1e-9 ), sr19.ot, sr19.sErr.aErr, sr19.sErr.rErr, sr19.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr20.name, sr20.et, ( sr20.flops * 1e-9 ), sr20.ot, sr20.sErr.aErr, sr20.sErr.rErr, sr20.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr21.name, sr21.et, ( sr21.flops * 1e-9 ), sr21.ot, sr21.sErr.aErr, sr21.sErr.rErr, sr21.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr22.name, sr22.et, ( sr22.flops * 1e-9 ), sr22.ot, sr22.sErr.aErr, sr22.sErr.rErr, sr22.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr23.name, sr23.et, ( sr23.flops * 1e-9 ), sr23.ot, sr23.sErr.aErr, sr23.sErr.rErr, sr23.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr24.name, sr24.et, ( sr24.flops * 1e-9 ), sr24.ot, sr24.sErr.aErr, sr24.sErr.rErr, sr24.sErr.pos ); fflush(stdout);
//	printf( "%25s %15.7lf %8.3lf %15.7lf %11.3le %13.3le %12d\n", sr25.name, sr25.et, ( sr25.flops * 1e-9 ), sr25.ot, sr25.sErr.aErr, sr25.sErr.rErr, sr25.sErr.pos ); fflush(stdout);

	return( EXIT_SUCCESS );
}


